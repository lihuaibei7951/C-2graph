#include "hip/hip_runtime.h"
#include <time.h>
#include "Util.cuh"
#include "DeviceMemory.cuh"

__device__ volatile int g_mutex1;
__device__ volatile int g_mutex2;
__device__ volatile int g_mutex3;
__device__ volatile int g_mutex4;
__device__ volatile int g_mutex5;
//原子操作重写
__device__ static float atomicMin(float *address, float val) {
	int *address_as_i = (int *) address;
	int old = *address_as_i, assumed;
	do {
		assumed = old;
		old = ::atomicCAS(address_as_i, assumed,
				__float_as_int(::fminf(val, __int_as_float(assumed))));
	} while (assumed != old);
	return __int_as_float(old);
}

//kernel_ update
//template<typename ValueType>
__global__ void CalcuSSSP(const Vertex *csr_v,const Vertex *csr_e, const ValueType *csr_w, ValueType *dis, 
		Vertex *active_vert, Vertex *active_vert_num, bool *isactive,  
		const Vertex vert_num, Vertex source,Vertex *iteration_id, Vertex *iteration_num ,Vertex *iteration_act_num);         
int main(int argc, char **argv) {
    // Initialize graph data in host & device memory
    hipFree(0);
    // 获取命令行参数
    std::string dir = argv[1];
    ofstream outfile(argv[2]);

    // 获取，csr_v ,csr_e ,v_r,degree,order;
    Graph graph(dir);
    
	DeviceMemory device_memory(graph.vert_num, graph.edge_num);

    device_memory.CudaMemcpyGraph(graph);
    std::cout << "test for study how to use cuda" << endl;
	int vert_num = graph.vert_num;
	int edge_num = graph.edge_num;


	ValueType *h_distance = new ValueType[vert_num];
	
	int *iteration_id;
	hipMalloc(&iteration_id, sizeof(int)*1000);

	
	
	int *iteration_num = new int[1];
	int *iteration_act_num = new int[1000];
	int source = 12;
	
	hipStream_t stream1;
	hipStreamCreate(&stream1);
	hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        cout << "\n==================== SSSP with FORWARD PUSH starts ============" <<endl;
	
	

	int cnt = 0;

	while (1) {
		if(cnt % 10 ==0){
		hipDeviceSynchronize();
		cout<<cnt<<endl;
		}
		hipDeviceSynchronize();
		cout<<cnt<<endl;
		CalcuSSSP<<<MAX_BLOCKS_NUM, THREADS_PER_BLOCK,0,stream1>>>(
				device_memory.csr_v, device_memory.csr_e, device_memory.csr_w, device_memory.distance, 
				device_memory.active_vert,device_memory.active_vert_num,device_memory.isactive,  
				vert_num, source,iteration_id,device_memory.iteration_num,device_memory.iteration_act_num);
		
		cnt++;
		
		if (cnt == 10) {
			break;
		}

	}
	

	hipEventRecord(stop);
     hipEventSynchronize(stop);
     CUDA_ERROR(hipMemcpy(iteration_num, device_memory.iteration_num, 
		sizeof(int)*1, hipMemcpyDeviceToHost));
	cout << "flag 已设置成 -1  终止条件以满足		iteration_num："<<iteration_num[0]<<endl;
	CUDA_ERROR(hipMemcpy(iteration_act_num, device_memory.iteration_act_num, 
		sizeof(int)*1000, hipMemcpyDeviceToHost));
	cout << "0	act_num：1"<<endl;
	for(int i = 1 ;iteration_act_num[i]!=0 ; i++){
		cout <<i<< "	act_num："<<iteration_act_num[i]<<endl;
		if(i>998) break;
	}
	CUDA_ERROR(hipMemcpy(h_distance, device_memory.distance, 
					vert_num*sizeof(ValueType), hipMemcpyDeviceToHost));
	for (int i = 0; i < vert_num; i++) {
				outfile<<  i << "\t" << h_distance[i] << endl;
	}
        float runtime = 0;
        hipEventElapsedTime(&runtime, start, stop);
        cout << "gpu runtime: " << runtime/1000.0<< " seconds" <<endl;
        cout << "源顶点source = " << source-1 <<endl;

	return 0;
}


//kernel_1 init

//kernel_2 update
//template<typename ValueType>
//template<typename ValueType>
__global__ void CalcuSSSP(const Vertex *csr_v,const Vertex *csr_e, const ValueType *csr_w, ValueType *dis, 
		Vertex *active_vert, Vertex *active_vert_num, bool *isactive,  
		const Vertex vert_num, Vertex source,Vertex *iteration_id, Vertex *iteration_num ,Vertex *iteration_act_num){        

	size_t thread_id = threadIdx.x;
	size_t schedule_offset_init = blockDim.x * blockIdx.x;
	size_t vid = 0;

	while (schedule_offset_init < vert_num) {

		vid = schedule_offset_init + thread_id;

		if (vid < vert_num ) {
			dis[vid] = 99999999;
			isactive[vid] = false;
		}
		if (vid < 1000) {
			iteration_id[vid] = vid;
		}
		schedule_offset_init += blockDim.x * gridDim.x;
	}

	//prepare for iteration
	size_t global_id = thread_id + blockDim.x * blockIdx.x;
	if (global_id == 0) {
		*active_vert_num = 1;
		active_vert[0] = source;
		dis[source] = 0;
		g_mutex1 = 0;
          g_mutex2 = 0;
          g_mutex4 = 0;
          g_mutex5 = 0;
	}
	__threadfence();
	if (threadIdx.x == 0) {
		atomicAdd((int*) &g_mutex3, 1);
		while ((g_mutex3 == 0) || (g_mutex3 % gridDim.x) ) {}
	}
	__syncthreads();



	int l_iteration_id = 0;
	int total_avtive_num = 1;
	size_t lane_id = thread_id % THREADS_PER_WARP;
	size_t warp_id = thread_id / THREADS_PER_WARP;

	typedef hipcub::BlockScan<int, THREADS_PER_BLOCK> BlockScan;	
	__shared__ typename BlockScan::TempStorage block_temp_storage;

        volatile __shared__ int comm[THREADS_PER_BLOCK/THREADS_PER_WARP][3];//[256/32][3]第一维是多少个warp，256/32=8，8个warp，
	volatile __shared__ ValueType commr[THREADS_PER_BLOCK/THREADS_PER_WARP];//每个warp对应一个值 
        volatile __shared__ int comm2[THREADS_PER_BLOCK]; //一维数组大小256，int
        volatile __shared__ int commd2[THREADS_PER_BLOCK]; //out-degree
	volatile __shared__ ValueType commr2[THREADS_PER_BLOCK];

	//while (!(l_iteration_id > 20 && *active_verts_num < 100)) {
	while (*active_vert_num > 0&&l_iteration_id<1000) {
		l_iteration_id += 1;
		__syncthreads();
		total_avtive_num = *active_vert_num;
		__threadfence();
		if (threadIdx.x == 0) {
			atomicAdd((int *)&g_mutex4, 1);
			while (g_mutex4 < gridDim.x * iteration_id[l_iteration_id]) {}
	     }
		__syncthreads();
		size_t schedule_offset = blockDim.x * blockIdx.x;
		size_t idx = 0;
		int row_start, row_end;
		int u, v;
		ValueType du, weight; //dis value of u

		while (schedule_offset < total_avtive_num) {
			idx = schedule_offset + thread_id;
			if (idx < total_avtive_num) {
				u = active_vert[idx];
				du = dis[u];
				row_start = csr_v[u];
				row_end = csr_v[u+1];

			} else {
				row_start = 0;
				row_end = 0;
			}	

			while (__syncthreads_or((row_end-row_start)>=THREADS_PER_BLOCK)) {
				if ((row_end-row_start) >= THREADS_PER_BLOCK) {
					comm[0][0] = thread_id; //I (thread_id) want to process the active vertex assigned to me
				}
				__syncthreads(); //all threads in one block vote to processing their own vertices

				if (comm[0][0] == thread_id) {
					comm[0][1] = row_start; //the vertx owned by me will be processed in this <1>-while loop.
					comm[0][2] = row_end;
					commr[0] = du;
					row_start = row_end;//avoid processing this vertex repeatedly in <2>&<3>-while
				}
				__syncthreads(); //all threads are ready to process the selected vertex

				size_t push_st = comm[0][1] + thread_id; //process the "push_st"-th outgoing edge at first.
				size_t push_ed = comm[0][2];

				while (__syncthreads_or(push_st<push_ed)) {
					if (push_st < push_ed) {
						v = csr_e[push_st]; //target vertex id
						weight = commr[0]+csr_w[push_st];
						ValueType old = atomicMin(dis+v, weight);
						if(old != dis[v])
							isactive[v] = true;

					}
					push_st += THREADS_PER_BLOCK;//直到u的所有外邻居被处理
				}

			}//while<0>, outdeg > 256



			////<2> warp(32)
			while (__any_sync(FULL_MASK, (row_end-row_start)>=THREADS_PER_WARP)) {
				if ((row_end-row_start) >= THREADS_PER_WARP) {
					comm[warp_id][0] = lane_id;
				}

				if (comm[warp_id][0] == lane_id) {
					comm[warp_id][1] = row_start; //vertex owned by the "lane_id"-th thread in a warp is scheduled
					comm[warp_id][2] = row_end;
					commr[warp_id] = du;
					row_start = row_end; //avoid processing this vertex repeatedly in <3>-while
				}
				size_t push_st = comm[warp_id][1] + lane_id; //process the "push_st"-th outgoing edge at first.
				size_t push_ed = comm[warp_id][2];

				////<2.1>
				while (__any_sync(FULL_MASK, push_st<push_ed)) {
					if (push_st < push_ed) {
						v = csr_e[push_st];
						weight = commr[warp_id] + csr_w[push_st];
						ValueType old = atomicMin(dis+v, weight);
						if(old != dis[v])
							isactive[v] = true;
					}
					push_st += THREADS_PER_WARP; //until all outgoing edges of "u" have been processed
				}//while<2>, 处理所有outdeg > 32
			}

			//then, the out-degree of "u" is less than THREADS_PER_WARP(32)	
			int thread_count = row_end - row_start;
			int deg = thread_count;
			int scatter = 0, total = 0;
			__syncthreads();
			BlockScan(block_temp_storage).ExclusiveSum(thread_count, scatter, total);
			__syncthreads();

			int progress = 0;

			///<3>
			while (progress < total) {
				int remain = total - progress;
				while (scatter<(progress+THREADS_PER_BLOCK) && (row_start<row_end)) {
					comm2[scatter-progress] = row_start;
					commd2[scatter-progress] = deg;
					commr2[scatter-progress] = du;
					scatter++;
					row_start++;
				}
				__syncthreads();
				int cur_batch_count = min(remain, (int)THREADS_PER_BLOCK);
				if (thread_id < cur_batch_count) {
					v = csr_e[comm2[thread_id]];
					weight = commr2[thread_id]+csr_w[comm2[thread_id]];
					ValueType old = atomicMin(dis+v, weight);
					if(old != dis[v])
						isactive[v] = true;
				}
				__syncthreads();
				progress += THREADS_PER_BLOCK;
			}
			schedule_offset += blockDim.x * gridDim.x;
		}
		__syncthreads();
		__threadfence();
		if (threadIdx.x == 0) {
			atomicAdd((int *) &g_mutex5, 1);
			while (g_mutex5 < gridDim.x * iteration_id[l_iteration_id]) {}
		}
		__syncthreads();
		__threadfence();
		*active_vert_num = 0;
		
		if (threadIdx.x == 0) {
			atomicAdd((int *) &g_mutex1, 1);
			while (g_mutex1 < gridDim.x * iteration_id[l_iteration_id]) {}
		}
		__syncthreads();

		volatile __shared__ size_t output_cta_offset;

		size_t thread_idx = threadIdx.x;
		size_t schedule_offset_barrir = blockDim.x * blockIdx.x;
		size_t vid = 0;
		while (__syncthreads_or(schedule_offset_barrir < vert_num)) {
			vid = schedule_offset_barrir + thread_idx;
			int thread_cnt = 0;
			if (vid < vert_num) {
				if (isactive[vid]) {
					isactive[vid] = false;
					thread_cnt = 1;
				}
			}
			int scatter = 0, total = 0;
			__syncthreads();
			BlockScan(block_temp_storage).ExclusiveSum(thread_cnt, scatter, total);
			__syncthreads();
			if (thread_id == 0) {
				output_cta_offset = atomicAdd(active_vert_num, total);
			}
			__syncthreads();
			if (thread_cnt > 0) {
				active_vert[output_cta_offset + scatter] = vid;
			}

			schedule_offset_barrir += blockDim.x * gridDim.x;                                 
		}
		__syncthreads();
		if (threadIdx.x == 0) {
			atomicAdd((int *)&g_mutex2, 1);
			while (g_mutex2 < gridDim.x * iteration_id[l_iteration_id]) {}
			
		}
		if (threadIdx.x == 0) {
			iteration_num[0] = l_iteration_id;
			if(l_iteration_id<1000)	iteration_act_num[l_iteration_id]= *active_vert_num;
			
		}
		__syncthreads();

	}
	
}                                

