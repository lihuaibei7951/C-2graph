
#include <hip/hip_runtime.h>
#include <stdio.h>

// 定义向量长度
#define N 10

// 核函数：向量相加
__global__ void vectorAddKernel(int *a, int *b, int *c) {
    // 获取当前线程的全局索引
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // 确保索引在向量长度范围内
    
    int iter = 0;
    while(iter<1){
    iter++;
    int x;
	if (tid < N) {
        // 将相应位置的元素相加，并将结果存储在 c 中
        x = b[tid];
        c[tid] = x;
    }

    }
    
}

int main(int argc, char **argv) {
    // 定义向量 a, b, c
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // 在设备上分配内存
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    // 初始化向量 a, b
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    // 将向量 a, b 复制到设备
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    // 定义块大小和网格大小
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    // 调用核函数
    vectorAddKernel<<<gridSize, blockSize>>>(dev_a, dev_b, dev_c);

    // 将结果复制回主机
    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // 打印结果
    printf("Vector addition result:\n");
    for (int i = 0; i < N; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // 释放设备上的内存
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}