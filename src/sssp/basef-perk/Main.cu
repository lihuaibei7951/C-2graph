#include "hip/hip_runtime.h"
#include <time.h>
#include "Util.cuh"
#include "DeviceMemory.cuh"
#include "SSSP.h"
#include <sys/time.h>

__device__ volatile int g_mutex1;
__device__ volatile int g_mutex2;
__device__ volatile int g_mutex3;
__device__ volatile int g_mutex4;
__device__ volatile int g_mutex5;


//原子操作重写
__device__ static float atomicMin(float *address, float val) {
	int *address_as_i = (int *) address;
	int old = *address_as_i, assumed;
	do {
		assumed = old;
		old = ::atomicCAS(address_as_i, assumed,
				__float_as_int(::fminf(val, __int_as_float(assumed))));
	} while (assumed != old);
	return __int_as_float(old);
}

//kernel_1 init
// 检查 CUDA 错误的宏
#define CUDA_CHECK(call)                                                      \
    do {                                                                      \
        hipError_t err = call;                                               \
        if (err != hipSuccess) {                                             \
            std::cerr << "CUDA error in " << __FILE__ << "@" << __LINE__      \
                      << ": " << hipGetErrorString(err) << std::endl;        \
            std::exit(EXIT_FAILURE);                                          \
        }                                                                     \
    } while (0)
//kernel_ update
//template<typename ValueType>
__global__ void CalcuSSSP(const Vertex *csr_v,const Vertex *csr_e, ValueType *csr_w, ValueType *dis,
                          Vertex *active_vert, Vertex *active_vert_num, bool *isactive,
                          const Vertex vert_num, Vertex source,Vertex *iteration_id,int iter);

int main(int argc, char **argv) {
    // Initialize graph data in host & device memory
    hipFree(0);
    // 获取命令行参数
    std::string dir = argv[1];
    int source = atoi(argv[2]);
    int iter = atoi(argv[3]);
    graph.Graphinit(dir);

    DeviceMemory device_memory(graph.vert_num, graph.edge_num);

    device_memory.CudaMemcpyGraph(graph);
    std::cout << "test for study how to use cuda" << endl;

    int *iteration_id;
    hipMalloc(&iteration_id, sizeof(int)*2000);


    BufferQueue *bufferqueue;
    bufferqueue = new BufferQueue;
    initQueue(bufferqueue, 20);//1000


    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    pthread_t Pthread;

    if(0 != pthread_create(&Pthread, NULL, SSSP_CPU, (void *)bufferqueue)){
        printf("Error: 线程创建失败\n");
        exit(-1);
    }


    cout << "\n==================== SSSP with FORWARD PUSH starts ============" <<endl;

    int cnt = 0;

    int *flagG;
    CUDA_ERROR(hipMalloc(&flagG, sizeof(int)));
    CUDA_ERROR(hipMemset(flagG, -1, sizeof(int)));

    struct timeval t_start, t_stop;
    double timeuse;
    gettimeofday(&t_start, NULL);


    using namespace std::chrono;

    // 存储时间点
    std::vector<high_resolution_clock::time_point> timestamps;

    // 记录第一个时间点
    timestamps.push_back(high_resolution_clock::now());
        hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);



    
    while (1) {
    while(graph.csr_v[source+1]-graph.csr_v[source]<1){//
            ++source;
        }

        CalcuSSSP<<<MAX_BLOCKS_NUM, THREADS_PER_BLOCK,0,stream2>>>(
                device_memory.csr_v, device_memory.csr_e, device_memory.csr_w, device_memory.distance2,
                device_memory.active_vert2,device_memory.active_vert_num2,device_memory.isactive,
                graph.vert_num, source++,iteration_id,iter);
        
        
        if(cnt){
       
         
            CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->distance, device_memory.distance1,
                                       graph.vert_num *sizeof(ValueType), hipMemcpyDeviceToHost, stream1 ));//1

            //额外开销

            CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->global_ft_cnt, device_memory.active_vert_num1,
                                       sizeof(int), hipMemcpyDeviceToHost, stream1));

            CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->global_ft, device_memory.active_vert1,
                                       graph.vert_num *sizeof(int), hipMemcpyDeviceToHost, stream1 ));

            CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->flag, flagG,
                                       sizeof(int), hipMemcpyDeviceToHost, stream1));

            ////更新队列信息
            hipStreamSynchronize(stream1);
            bufferqueue->front->source = source-1;
            bufferqueue->front = bufferqueue->front->next; // 指针后移
            bufferqueue->length++;
            cnt++;
        }

while(graph.csr_v[source+1]-graph.csr_v[source]<1){//
            ++source;
        }
        CalcuSSSP<<<MAX_BLOCKS_NUM, THREADS_PER_BLOCK,0,stream1>>>(
                device_memory.csr_v, device_memory.csr_e, device_memory.csr_w, device_memory.distance1,
                device_memory.active_vert1,device_memory.active_vert_num1,device_memory.isactive,
                graph.vert_num, source++,iteration_id,iter);

        CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->distance, device_memory.distance2,
                                   graph.vert_num*sizeof(ValueType), hipMemcpyDeviceToHost, stream2));

        //额外开销

        CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->global_ft_cnt, device_memory.active_vert_num2,
                                   sizeof(int), hipMemcpyDeviceToHost, stream2));

        CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->global_ft, device_memory.active_vert2,
                                   graph.vert_num *sizeof(int), hipMemcpyDeviceToHost, stream2 ));

        CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->flag, flagG,
                                   sizeof(int), hipMemcpyDeviceToHost, stream2));

        ////更新队列信息
        // Stream2 完成后，更新队列
        hipStreamSynchronize(stream2);
        bufferqueue->front->source = source-1;
        bufferqueue->front = bufferqueue->front->next; // 指针后移
        bufferqueue->length++;
        cnt++;
          timestamps.push_back(high_resolution_clock::now());
         cout << "当前bufferqueue长度为:\t" << bufferqueue->length << endl;


        if (cnt >= 2) {
                  
         
            CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->distance, device_memory.distance1,
                                       graph.vert_num *sizeof(ValueType), hipMemcpyDeviceToHost, stream1 ));//1

            //额外开销

            CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->global_ft_cnt, device_memory.active_vert_num1,
                                       sizeof(int), hipMemcpyDeviceToHost, stream1));

            CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->global_ft, device_memory.active_vert1,
                                       graph.vert_num *sizeof(int), hipMemcpyDeviceToHost, stream1 ));

            CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->flag, flagG,
                                       sizeof(int), hipMemcpyDeviceToHost, stream1));

            ////更新队列信息
            hipStreamSynchronize(stream1);
            bufferqueue->front->source = source-2;
            bufferqueue->front = bufferqueue->front->next; // 指针后移
            bufferqueue->length++;
            cnt++;
            hipDeviceSynchronize();
            bufferqueue->flag = -1;
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            break;
        }

    }


    
    float runtime = 0;
    hipEventElapsedTime(&runtime, start, stop);
    

    pthread_join(Pthread, NULL);
    cout << "gpu runtime: " << runtime/1000.0<< " seconds" <<endl;
    cout << "源顶点source = " << source <<endl;

    gettimeofday(&t_stop, NULL);
    timeuse = (t_stop.tv_sec - t_start.tv_sec) + (double)(t_stop.tv_usec - t_start.tv_usec)/1000000.0;
    cout << "main total timeval runtime: " << timeuse << " seconds" << endl;

 // 计算并输出时间差
    for (size_t i = 1; i < timestamps.size(); ++i) {
        auto duration = duration_cast<milliseconds>(timestamps[i] - timestamps[0]);
        std::cout << duration.count()  << std::endl;
    }

    return 0;
}


//kernel_2 update
//template<typename ValueType>
//template<typename ValueType>
__global__ void CalcuSSSP(const Vertex *csr_v,const Vertex *csr_e, ValueType *csr_w, ValueType *dis,
                          Vertex *active_vert, Vertex *active_vert_num, bool *isactive,
                          const Vertex vert_num, Vertex source,Vertex *iteration_id,int iter){

    size_t thread_id = threadIdx.x;
    size_t schedule_offset_init = blockDim.x * blockIdx.x;
    size_t vid = 0;

    while (schedule_offset_init < vert_num) {

        vid = schedule_offset_init + thread_id;

        if (vid < vert_num ) {
            dis[vid] = 99999999.0;
            isactive[vid] = false;
        }
        if (vid < 2000) {
            iteration_id[vid] = vid;
        }
        schedule_offset_init += blockDim.x * gridDim.x;
    }

    //prepare for iteration
    size_t global_id = thread_id + blockDim.x * blockIdx.x;
    if (global_id == 0) {
        *active_vert_num = 1;
        active_vert[0] = source;
        dis[source] = 0;
        g_mutex1 = 0;
        g_mutex2 = 0;
        g_mutex4 = 0;
        g_mutex5 = 0;
    }
    __threadfence();
    if (threadIdx.x == 0) {
        atomicAdd((int*) &g_mutex3, 1);
        while ((g_mutex3 == 0) || (g_mutex3 % gridDim.x) ) {}
    }
    __syncthreads();



    int l_iteration_id = 0;
    int total_avtive_num = 1;
    size_t lane_id = thread_id % THREADS_PER_WARP;
    size_t warp_id = thread_id / THREADS_PER_WARP;

    typedef hipcub::BlockScan<int, THREADS_PER_BLOCK> BlockScan;
    __shared__ typename BlockScan::TempStorage block_temp_storage;

    volatile __shared__ int comm[THREADS_PER_BLOCK/THREADS_PER_WARP][3];//[256/32][3]第一维是多少个warp，256/32=8，8个warp，
    volatile __shared__ ValueType commr[THREADS_PER_BLOCK/THREADS_PER_WARP];//每个warp对应一个值
    volatile __shared__ int comm2[THREADS_PER_BLOCK]; //一维数组大小256，int
    volatile __shared__ int commd2[THREADS_PER_BLOCK]; //out-degree
    volatile __shared__ ValueType commr2[THREADS_PER_BLOCK];

    //while (!(l_iteration_id > 20 && *active_verts_num < 10)) {
     while ((*active_vert_num > 0&&l_iteration_id<iter)&&(l_iteration_id <10||*active_vert_num>100) ){
        //while (l_iteration_id < 1) {
        l_iteration_id += 1;
        __syncthreads();
        if (threadIdx.x == 0) {
            atomicAdd((int *)&g_mutex4, 1);
            while (g_mutex4 < gridDim.x * iteration_id[l_iteration_id]) {}
        }
        __syncthreads();
        size_t schedule_offset = blockDim.x * blockIdx.x;
        size_t idx = 0;
        int row_start, row_end;
        int u, v;
        ValueType du, weight; //dis value of u

        while (schedule_offset < *active_vert_num) {
            idx = schedule_offset + thread_id;
            if (idx < *active_vert_num) {
                u = active_vert[idx];
                du = dis[u];
                row_start = csr_v[u];
                row_end = csr_v[u+1];

            } else {
                row_start = 0;
                row_end = 0;
            }

            while (__syncthreads_or((row_end-row_start)>=THREADS_PER_BLOCK)) {
                if ((row_end-row_start) >= THREADS_PER_BLOCK) {
                    comm[0][0] = thread_id; //I (thread_id) want to process the active vertex assigned to me
                }
                __syncthreads(); //all threads in one block vote to processing their own vertices

                if (comm[0][0] == thread_id) {
                    comm[0][1] = row_start; //the vertx owned by me will be processed in this <1>-while loop.
                    comm[0][2] = row_end;
                    commr[0] = du;
                    row_start = row_end;//avoid processing this vertex repeatedly in <2>&<3>-while
                }
                __syncthreads(); //all threads are ready to process the selected vertex

                size_t push_st = comm[0][1] + thread_id; //process the "push_st"-th outgoing edge at first.
                size_t push_ed = comm[0][2];

                while (__syncthreads_or(push_st<push_ed)) {
                    if (push_st < push_ed) {
                        v = csr_e[push_st]; //target vertex id
                        weight = commr[0]+csr_w[push_st];
                        ValueType old = atomicMin(dis+v, weight);
                        if(old != dis[v])
                            isactive[v] = true;

                    }
                    push_st += THREADS_PER_BLOCK;//直到u的所有外邻居被处理
                }

            }//while<0>, outdeg > 256

            ////<2> warp(32)
            while (__any_sync(FULL_MASK, (row_end-row_start)>=THREADS_PER_WARP)) {
                if ((row_end-row_start) >= THREADS_PER_WARP) {
                    comm[warp_id][0] = lane_id;
                }

                if (comm[warp_id][0] == lane_id) {
                    comm[warp_id][1] = row_start; //vertex owned by the "lane_id"-th thread in a warp is scheduled
                    comm[warp_id][2] = row_end;
                    commr[warp_id] = du;
                    row_start = row_end; //avoid processing this vertex repeatedly in <3>-while
                }
                size_t push_st = comm[warp_id][1] + lane_id; //process the "push_st"-th outgoing edge at first.
                size_t push_ed = comm[warp_id][2];

                ////<2.1>
                while (__any_sync(FULL_MASK, push_st<push_ed)) {
                    if (push_st < push_ed) {
                        v = csr_e[push_st];
                        weight = commr[warp_id] + csr_w[push_st];
                        ValueType old = atomicMin(dis+v, weight);
                        if(old != dis[v])
                            isactive[v] = true;
                    }
                    push_st += THREADS_PER_WARP; //until all outgoing edges of "u" have been processed
                }//while<2>, 处理所有outdeg > 32
            }

            //then, the out-degree of "u" is less than THREADS_PER_WARP(32)
            int thread_count = row_end - row_start;
            int deg = thread_count;
            int scatter = 0, total = 0;
            __syncthreads();
            BlockScan(block_temp_storage).ExclusiveSum(thread_count, scatter, total);
            __syncthreads();

            int progress = 0;

            ///<3>
            while (progress < total) {
                int remain = total - progress;
                while (scatter<(progress+THREADS_PER_BLOCK) && (row_start<row_end)) {
                    comm2[scatter-progress] = row_start;
                    commd2[scatter-progress] = deg;
                    commr2[scatter-progress] = du;
                    scatter++;
                    row_start++;
                }
                __syncthreads();
                int cur_batch_count = min(remain, (int)THREADS_PER_BLOCK);
                if (thread_id < cur_batch_count) {
                    v = csr_e[comm2[thread_id]];
                    weight = commr2[thread_id]+csr_w[comm2[thread_id]];
                    ValueType old = atomicMin(dis+v, weight);
                    if(old != dis[v])
                        isactive[v] = true;
                }
                __syncthreads();
                progress += THREADS_PER_BLOCK;
            }
            schedule_offset += blockDim.x * gridDim.x;
        }
        __syncthreads();
 __syncthreads();
        if (threadIdx.x == 0) {
            atomicAdd((int *)&g_mutex5, 1);
            while (g_mutex5 < gridDim.x * iteration_id[l_iteration_id]) {}
        }
        __syncthreads();
        *active_vert_num = 0;

        __threadfence();
        if (threadIdx.x == 0) {
            atomicAdd((int *) &g_mutex1, 1);
            while (g_mutex1 < gridDim.x * iteration_id[l_iteration_id]) {}
        }
        __syncthreads();

        volatile __shared__ size_t output_cta_offset;

        size_t thread_idx = threadIdx.x;
        size_t schedule_offset_barrir = blockDim.x * blockIdx.x;
        size_t vid = 0;
        while (__syncthreads_or(schedule_offset_barrir < vert_num)) {
            vid = schedule_offset_barrir + thread_idx;
            int thread_cnt = 0;
            if (vid < vert_num) {
                if (isactive[vid]) {
                    isactive[vid] = false;
                    thread_cnt = 1;
                }
            }
            int scatter = 0, total = 0;
            __syncthreads();
            BlockScan(block_temp_storage).ExclusiveSum(thread_cnt, scatter, total);
            __syncthreads();
            if (thread_id == 0) {
                output_cta_offset = atomicAdd(active_vert_num, total);
            }
            __syncthreads();
            if (thread_cnt > 0) {
                active_vert[output_cta_offset+scatter] = vid;
            }

            schedule_offset_barrir += blockDim.x * gridDim.x;
        }
        __syncthreads();
        if (threadIdx.x == 0) {
            atomicAdd((int *)&g_mutex2, 1);
            while (g_mutex2 < gridDim.x * iteration_id[l_iteration_id]) {}

        }
        total_avtive_num = *active_vert_num;
        __syncthreads();
    }

}

