#include "hip/hip_runtime.h"
#include <time.h>
#include "Util.cuh"
#include "DeviceMemory.cuh"

__device__ volatile int g_mutex1;
__device__ volatile int g_mutex2;
__device__ volatile int g_mutex3;
__device__ volatile int g_mutex4;
__device__ volatile int g_mutex5;

//原子操作重写
__device__ static float atomicMin(float *address, float val) {
    int *address_as_i = (int *) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
                          __float_as_int(::fminf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

//kernel_ update
template<typename ValueType>
__global__ void CalcuSSSP(const Vertex *csr_v, const Vertex *csr_e, const ValueType *csr_w, ValueType *dis,
                          const Vertex *csr_ov, const Vertex *csr_idx,
                          Vertex *active_vert1,Vertex *active_vert2, Vertex *active_vert_num1,Vertex *active_vert_num2, bool *isactive,
                          const Vertex vert_num, Vertex source, Vertex *iteration_id, Vertex *iteration_num,
                          Vertex *iteration_act_num);

int main(int argc, char **argv) {
    // Initialize graph data in host & device memory
    hipFree(0);
    // 获取命令行参数
    std::string dir = argv[1];

    // 获取，csr_v ,csr_e ,v_r,degree,order;
    Graph graph(dir);

    DeviceMemory device_memory(graph.vert_num, graph.rule_num,graph.edge_num,graph.w_num);

    device_memory.CudaMemcpyGraph(graph);
    std::cout << "test for study how to use cuda" << endl;
    int vert_num = graph.vert_num;
    int add_num = graph.add_num;
    int rule_num = graph.rule_num;
    int edge_num = graph.edge_num;

    ValueType *h_distance = new ValueType[vert_num];

    int *iteration_id;
    iteration_id = NULL;
    hipMalloc(&iteration_id, sizeof(int) * 2000);

    int *iteration_num = new int[1];
    int *ac1 = new int[1];
    int *ac2 = new int[1];
    int *iteration_act_num = new int[2000];
    int source = 121;

    hipStream_t stream1;
    hipStreamCreate(&stream1);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    cout << "\n==================== SSSP with FORWARD PUSH starts ============" << endl;

    int cnt = 0;

    while (1) {
        if(cnt % 10 ==0){
		hipDeviceSynchronize();
		cout<<cnt<<endl;
		}
		
		CUDA_ERROR(hipMemset(device_memory.iteration_num, 0, sizeof(int)));
            CalcuSSSP<<<MAX_BLOCKS_NUM, THREADS_PER_BLOCK,0,stream1>>>(
                    device_memory.csr_v, device_memory.csr_e, device_memory.csr_w, device_memory.distance,
                    device_memory.csr_ov, device_memory.csr_idx,
                    device_memory.active_vert1,device_memory.active_vert2, device_memory.active_vert_num1,device_memory.active_vert_num2, device_memory.isactive,
                    vert_num, source, iteration_id, device_memory.iteration_num, device_memory.iteration_act_num);


        cnt++;

        if (cnt == 1) {
        	hipDeviceSynchronize();
            break;
        }

    }


    hipEventRecord(stop);
    hipEventSynchronize(stop);

    //cout << "flag 已设置成 -1  终止条件以满足		iteration_num：" << endl;
    
    CUDA_ERROR(hipMemcpy(iteration_num, device_memory.iteration_num, sizeof(int) * 1, hipMemcpyDeviceToHost));
    
   /* cout << "flag 已设置成 -1  终止条件以满足		iteration_num：" << iteration_num[0] << endl;
    CUDA_ERROR(hipMemcpy(iteration_act_num, device_memory.iteration_act_num,sizeof(int) * 2000, hipMemcpyDeviceToHost));
    cout << "0	act_num：1" << endl;
    for (int i = 1; iteration_act_num[i] != 0; i++) {
        cout << i << "	act_num：" << iteration_act_num[i] << endl;
        if (i > 1980) break;
    }
    CUDA_ERROR(hipMemcpy(h_distance, device_memory.distance,
                          vert_num * sizeof(ValueType), hipMemcpyDeviceToHost));
    for (int i = 0; i < 100; i++) {
        cout << i << "\t" << h_distance[i] << endl;
    }*/
     cout << "\n==================== SSSP with FORWARD PUSH ends ============" << endl;
    float runtime = 0;
    hipEventElapsedTime(&runtime, start, stop);
    cout << "gpu runtime: " << runtime / 1000.0 << " seconds" << endl;
    cout << "源顶点source = " << source << endl;

    return 0;
}


//kernel_1 init
//kernel_2 update
//template<typename ValueType>
template<typename ValueType>
__global__ void CalcuSSSP(const Vertex *csr_v, const Vertex *csr_e, const ValueType *csr_w, ValueType *dis,
                          const Vertex *csr_ov, const Vertex *csr_idx,
                          Vertex *active_vert1,Vertex *active_vert2, Vertex *active_vert_num1,Vertex *active_vert_num2, bool *isactive,
                          const Vertex vert_num, Vertex source, Vertex *iteration_id, Vertex *iteration_num,
                          Vertex *iteration_act_num) {
    size_t thread_id = threadIdx.x;
    size_t schedule_offset_init = blockDim.x * blockIdx.x;
    size_t vid = 0;

    while (schedule_offset_init < vert_num) {

        vid = schedule_offset_init + thread_id;

        if (vid < vert_num) {
            dis[vid] = 99999999;
            isactive[vid] = false;
        }
        if (vid < 2000) {
            iteration_id[vid] = vid;
        }
        schedule_offset_init += blockDim.x * gridDim.x;
    }

    //prepare for iteration
    size_t global_id = thread_id + blockDim.x * blockIdx.x;
    if (global_id == 0) {
        active_vert1[0] = source;
        active_vert1[1] = source;
        active_vert1[2] = csr_ov[source];
        dis[source] = 0;
        *active_vert_num1 = 3;
        *active_vert_num2 = 0;
        g_mutex1 = 0;
        g_mutex2 = 0;
        g_mutex4 = 0;
        g_mutex5 = 0;
    }



    __threadfence();
    if (threadIdx.x == 0) {
        atomicAdd((int*) &g_mutex3, 1);
        while ((g_mutex3 == 0) || (g_mutex3 % gridDim.x) ) {}
    }
    __syncthreads();
    int l_iteration_id = 0;
    int total_avtive_num1;
    thread_id = threadIdx.x;//当前块内的线程id
    size_t lane_id = thread_id % THREADS_PER_WARP;
    size_t warp_id = thread_id / THREADS_PER_WARP;

    typedef hipcub::BlockScan<int, THREADS_PER_BLOCK> BlockScan;
    __shared__ typename BlockScan::TempStorage block_temp_storage;

    volatile __shared__ int comm[THREADS_PER_BLOCK / THREADS_PER_WARP][3];//[256/32][3]第一维是多少个warp，256/32=8，8个warp，
    volatile __shared__ ValueType commr[THREADS_PER_BLOCK / THREADS_PER_WARP];//每个warp对应一个值
    volatile __shared__ int commi[THREADS_PER_BLOCK / THREADS_PER_WARP][2];//每个warp对应3个值
    volatile __shared__ int comm2[THREADS_PER_BLOCK]; //一维数组大小256，int
    volatile __shared__ int commd2[THREADS_PER_BLOCK]; //out-degree
    volatile __shared__ ValueType commr2[THREADS_PER_BLOCK];
    volatile __shared__ int commi2[THREADS_PER_BLOCK][2];//每个warp对应3个值

    while ((*active_vert_num1 >0||*active_vert_num2 >0)&&l_iteration_id<2000) {
        l_iteration_id += 1;
        __syncthreads();

        if(l_iteration_id%2==1){
            total_avtive_num1 = *active_vert_num1/3;
            __syncthreads();

            size_t schedule_offset = blockDim.x * blockIdx.x;
            size_t idx=0;
                int row_start=0, row_end=0;
                int u=0, v=0, root=0, idxx=0;
                ValueType du=0, weight=0; //dis value of u


            while (schedule_offset < total_avtive_num1) {
                idx = schedule_offset + thread_id;
                if (idx < total_avtive_num1) {
                    u = active_vert1[idx * 3];//需要记录
                    root = active_vert1[idx * 3 + 1];//需要记录
                    idxx = active_vert1[idx * 3 + 2];//需要记录
                    du = dis[root];
                    row_start = csr_v[u];
                    row_end = csr_v[u + 1];

                } else {
                    row_start = 0;
                    row_end = 0;
                }

                while (__syncthreads_or((row_end - row_start) >= THREADS_PER_BLOCK)) {
                    if ((row_end - row_start) >= THREADS_PER_BLOCK) {
                        comm[0][0] = thread_id; //I (thread_id) want to process the active vertex assigned to me
                    }
                    __syncthreads(); //all threads in one block vote to processing their own vertices

                    if (comm[0][0] == thread_id) {
                        comm[0][1] = row_start; //the vertx owned by me will be processed in this <1>-while loop.
                        comm[0][2] = row_end;
                        commr[0] = du;
                        commi[0][0] = root;
                        commi[0][1] = idxx;
                        row_start = row_end;//avoid processing this vertex repeatedly in <2>&<3>-while
                    }
                    __syncthreads(); //all threads are ready to process the selected vertex

                    size_t push_st = comm[0][1] + thread_id; //process the "push_st"-th outgoing edge at first.
                    size_t push_ed = comm[0][2];

                    while (__syncthreads_or(push_st < push_ed)) {
                        if (push_st < push_ed) {
                            v = csr_e[push_st]; //target vertex id
                            int testx = commi[0][1] + csr_idx[push_st];
                            if(v<vert_num){
                                weight = commr[0] + csr_w[testx];
                                ValueType old = atomicMin(dis + v, weight);
                                if ((old != dis[v])&&!isactive[v]) {
                                    isactive[v] = true;
                                    int cur = atomicAdd(active_vert_num2, 3);
                                    active_vert2[cur] = v;
                                    active_vert2[cur+1] = v;
                                    active_vert2[cur+2] = csr_ov[v];
                                    //printf("1234.....\n");
                                }
                            } else {
                                int cur = atomicAdd(active_vert_num2,3);
                                active_vert2[cur]=v;
                                active_vert2[cur+1]=commi[0][0];
                                active_vert2[cur+2]=testx;
                            }
                        }
                        push_st += THREADS_PER_BLOCK;//直到u的所有外邻居被处理
                    }

                }//while<0>, outdeg > 256

                ////<2> warp(32)
                while (__any_sync(FULL_MASK, (row_end - row_start) >= THREADS_PER_WARP)) {
                    if ((row_end - row_start) >= THREADS_PER_WARP) {
                        comm[warp_id][0] = lane_id;
                    }

                    if (comm[warp_id][0] == lane_id) {
                        comm[warp_id][1] = row_start; //vertex owned by the "lane_id"-th thread in a warp is scheduled
                        comm[warp_id][2] = row_end;
                        commr[warp_id] = du;
                        commi[warp_id][0] = root;
                        commi[warp_id][1] = idxx;
                        row_start = row_end; //avoid processing this vertex repeatedly in <3>-while
                    }
                    size_t push_st = comm[warp_id][1] + lane_id; //process the "push_st"-th outgoing edge at first.
                    size_t push_ed = comm[warp_id][2];

                    ////<2.1>
                    while (__any_sync(FULL_MASK, push_st < push_ed)) {
                        if (push_st < push_ed) {
                            v = csr_e[push_st];
                            int testx = commi[warp_id][1] + csr_idx[push_st];
                            if(v<vert_num){

                                weight = commr[warp_id] + csr_w[testx];
                                ValueType old = atomicMin(dis + v, weight);
                                if (old != dis[v]&&!isactive[v]) {
                                    isactive[v] = true;
                                    int cur = atomicAdd(active_vert_num2, 3);
                                    active_vert2[cur] = v;
                                    active_vert2[cur+1] = v;
                                    active_vert2[cur+2] = csr_ov[v];
                                }
                            } else {
                                int cur = atomicAdd(active_vert_num2,3);
                                active_vert2[cur]=v;
                                active_vert2[cur+1]=commi[warp_id][0];
                                active_vert2[cur+2]=testx;

                            }

                        }
                        push_st += THREADS_PER_WARP; //until all outgoing edges of "u" have been processed
                    }//while<2>, 处理所有outdeg > 32
                }


                //then, the out-degree of "u" is less than THREADS_PER_WARP(32)
                int thread_count = row_end - row_start;
                int deg = thread_count;
                int scatter = 0, total = 0;
                __syncthreads();
                BlockScan(block_temp_storage).ExclusiveSum(thread_count, scatter, total);
                __syncthreads();

                int progress = 0;

                ///<3>
                while (progress < total) {
                    int remain = total - progress;
                    while (scatter < (progress + THREADS_PER_BLOCK) && (row_start < row_end)) {
                        comm2[scatter - progress] = row_start;
                        commd2[scatter - progress] = deg;
                        commr2[scatter - progress] = du;
                        commi2[scatter - progress][0] = root;
                        commi2[scatter - progress][1] = idxx;
                        scatter++;
                        row_start++;
                    }
                    __syncthreads();
                    int cur_batch_count = min(remain, (int) THREADS_PER_BLOCK);
                    if (thread_id < cur_batch_count) {
                        v = csr_e[comm2[thread_id]];
                        int testx = commi2[thread_id][1] + csr_idx[comm2[thread_id]];
                        if(v < vert_num){
                            weight = commr2[thread_id] + csr_w[testx];
                            ValueType old = atomicMin(dis + v, weight);
                            if (old != dis[v]&&!isactive[v]) {
                                isactive[v] = true;
                                int cur = atomicAdd(active_vert_num2, 3);
                                active_vert2[cur] = v;
                                active_vert2[cur+1] = v;
                                active_vert2[cur+2] = csr_ov[v];
                            }
                        }
                        else {
                            int cur = atomicAdd(active_vert_num2,3);
                            active_vert2[cur]=v;
                            active_vert2[cur+1]=commi2[thread_id][0];
                            active_vert2[cur+2]=testx;

                        }
                    }
                    __syncthreads();
                    progress += THREADS_PER_BLOCK;
                }
                schedule_offset += blockDim.x * gridDim.x;
            }
            __syncthreads();
            __threadfence();
            if (threadIdx.x == 0) {
                atomicAdd((int *)&g_mutex5, 1);
                while (g_mutex5 < gridDim.x * iteration_id[l_iteration_id]) {}

            }
            __syncthreads();
            *active_vert_num1 = 0;

            __threadfence();
            if (threadIdx.x == 0) {
                atomicAdd((int *) &g_mutex1, 1);
                while (g_mutex1 < gridDim.x * iteration_id[l_iteration_id]) {}
            }

            if (threadIdx.x == 0) {
                iteration_num[0] = l_iteration_id;
                if(l_iteration_id<2000)	iteration_act_num[l_iteration_id]= *active_vert_num2/3;
            }
            __syncthreads();

            size_t thread_idx = threadIdx.x;
            size_t schedule_offset_barrir = blockDim.x * blockIdx.x;
            size_t vid = 0;
            while (__syncthreads_or(schedule_offset_barrir < vert_num)) {
                vid = schedule_offset_barrir + thread_idx;
                if (vid < vert_num) {
                    if (isactive[vid]) {
                        isactive[vid] = false;
                    }
                }
                __syncthreads();

                schedule_offset_barrir += blockDim.x * gridDim.x;
            }

            __syncthreads();
            __threadfence();
            if (threadIdx.x == 0) {
                atomicAdd((int *)&g_mutex2, 1);
                while (g_mutex2 < gridDim.x * iteration_id[l_iteration_id]) {}

            }
            __syncthreads();

        }else{
            total_avtive_num1 = *active_vert_num2/3;
            __syncthreads();

            size_t schedule_offset = blockDim.x * blockIdx.x;
            size_t idx=0;
                int row_start=0, row_end=0;
                int u=0, v=0, root=0, idxx=0;
                ValueType du=0.0, weight=0.0; //dis value of u


            while (schedule_offset < total_avtive_num1) {
                idx = schedule_offset + thread_id;
                if (idx < total_avtive_num1) {
                    u = active_vert2[idx * 3];//需要记录
                    root = active_vert2[idx * 3 + 1];//需要记录
                    idxx = active_vert2[idx * 3 + 2];//需要记录
                    du = dis[root];
                    row_start = csr_v[u];
                    row_end = csr_v[u + 1];

                } else {
                    row_start = 0;
                    row_end = 0;
                }

                while (__syncthreads_or((row_end - row_start) >= THREADS_PER_BLOCK)) {
                    if ((row_end - row_start) >= THREADS_PER_BLOCK) {
                        comm[0][0] = thread_id; //I (thread_id) want to process the active vertex assigned to me
                    }
                    __syncthreads(); //all threads in one block vote to processing their own vertices

                    if (comm[0][0] == thread_id) {
                        comm[0][1] = row_start; //the vertx owned by me will be processed in this <1>-while loop.
                        comm[0][2] = row_end;
                        commr[0] = du;
                        commi[0][0] = root;
                        commi[0][1] = idxx;
                        row_start = row_end;//avoid processing this vertex repeatedly in <2>&<3>-while
                    }
                    __syncthreads(); //all threads are ready to process the selected vertex

                    size_t push_st = comm[0][1] + thread_id; //process the "push_st"-th outgoing edge at first.
                    size_t push_ed = comm[0][2];

                    while (__syncthreads_or(push_st < push_ed)) {
                        if (push_st < push_ed) {
                            v = csr_e[push_st]; //target vertex id
                            int testx = commi[0][1] + csr_idx[push_st];
                            if(v<vert_num){
                                weight = commr[0] + csr_w[testx];
                                ValueType old = atomicMin(dis + v, weight);
                                if ((old != dis[v])&&!isactive[v]) {
                                    isactive[v] = true;
                                    int cur = atomicAdd(active_vert_num1, 3);
                                    active_vert1[cur] = v;
                                    active_vert1[cur+1] = v;
                                    active_vert1[cur+2] = csr_ov[v];
                                    //printf("1234.....\n");
                                }
                            } else {
                                int cur = atomicAdd(active_vert_num1,3);
                                active_vert1[cur]=v;
                                active_vert1[cur+1]=commi[0][0];
                                active_vert1[cur+2]=testx;
                            }
                        }
                        push_st += THREADS_PER_BLOCK;//直到u的所有外邻居被处理
                    }

                }//while<0>, outdeg > 256

                ////<2> warp(32)
                while (__any_sync(FULL_MASK, (row_end - row_start) >= THREADS_PER_WARP)) {
                    if ((row_end - row_start) >= THREADS_PER_WARP) {
                        comm[warp_id][0] = lane_id;
                    }

                    if (comm[warp_id][0] == lane_id) {
                        comm[warp_id][1] = row_start; //vertex owned by the "lane_id"-th thread in a warp is scheduled
                        comm[warp_id][2] = row_end;
                        commr[warp_id] = du;
                        commi[warp_id][0] = root;
                        commi[warp_id][1] = idxx;
                        row_start = row_end; //avoid processing this vertex repeatedly in <3>-while
                    }
                    size_t push_st = comm[warp_id][1] + lane_id; //process the "push_st"-th outgoing edge at first.
                    size_t push_ed = comm[warp_id][2];

                    ////<2.1>
                    while (__any_sync(FULL_MASK, push_st < push_ed)) {
                        if (push_st < push_ed) {
                            v = csr_e[push_st];
                            int testx = commi[warp_id][1] + csr_idx[push_st];
                            if(v<vert_num){

                                weight = commr[warp_id] + csr_w[testx];
                                ValueType old = atomicMin(dis + v, weight);
                                if (old != dis[v]&&!isactive[v]) {
                                    isactive[v] = true;
                                    int cur = atomicAdd(active_vert_num1, 3);
                                    active_vert1[cur] = v;
                                    active_vert1[cur+1] = v;
                                    active_vert1[cur+2] = csr_ov[v];
                                }
                            } else {
                                int cur = atomicAdd(active_vert_num1,3);
                                active_vert1[cur]=v;
                                active_vert1[cur+1]=commi[warp_id][0];
                                active_vert1[cur+2]=testx;

                            }

                        }
                        push_st += THREADS_PER_WARP; //until all outgoing edges of "u" have been processed
                    }//while<2>, 处理所有outdeg > 32
                }


                //then, the out-degree of "u" is less than THREADS_PER_WARP(32)
                int thread_count = row_end - row_start;
                int deg = thread_count;
                int scatter = 0, total = 0;
                __syncthreads();
                BlockScan(block_temp_storage).ExclusiveSum(thread_count, scatter, total);
                __syncthreads();

                int progress = 0;

                ///<3>
                while (progress < total) {
                    int remain = total - progress;
                    while (scatter < (progress + THREADS_PER_BLOCK) && (row_start < row_end)) {
                        comm2[scatter - progress] = row_start;
                        commd2[scatter - progress] = deg;
                        commr2[scatter - progress] = du;
                        commi2[scatter - progress][0] = root;
                        commi2[scatter - progress][1] = idxx;
                        scatter++;
                        row_start++;
                    }
                    __syncthreads();
                    int cur_batch_count = min(remain, (int) THREADS_PER_BLOCK);
                    if (thread_id < cur_batch_count) {
                        v = csr_e[comm2[thread_id]];
                        int testx = commi2[thread_id][1] + csr_idx[comm2[thread_id]];
                        if(v < vert_num){
                            weight = commr2[thread_id] + csr_w[testx];
                            ValueType old = atomicMin(dis + v, weight);
                            if (old != dis[v]&&!isactive[v]) {
                                isactive[v] = true;
                                int cur = atomicAdd(active_vert_num1, 3);
                                active_vert1[cur] = v;
                                active_vert1[cur+1] = v;
                                active_vert1[cur+2] = csr_ov[v];
                            }
                        }
                        else {
                            int cur = atomicAdd(active_vert_num1,3);
                            active_vert1[cur]=v;
                            active_vert1[cur+1]=commi2[thread_id][0];
                            active_vert1[cur+2]=testx;

                        }
                    }
                    __syncthreads();
                    progress += THREADS_PER_BLOCK;
                }
                schedule_offset += blockDim.x * gridDim.x;
            }
            __syncthreads();
           __threadfence();
            if (threadIdx.x == 0) {
                atomicAdd((int *)&g_mutex5, 1);
                while (g_mutex5 < gridDim.x * iteration_id[l_iteration_id]) {}

            }
            __syncthreads();
            *active_vert_num2 = 0;

            __threadfence();
            if (threadIdx.x == 0) {
                atomicAdd((int *) &g_mutex1, 1);
                while (g_mutex1 < gridDim.x * iteration_id[l_iteration_id]) {}
            }

            if (threadIdx.x == 0) {
                iteration_num[0] = l_iteration_id;
                if(l_iteration_id<2000)	iteration_act_num[l_iteration_id]= *active_vert_num1/3;
            }
            __syncthreads();


            size_t thread_idx = threadIdx.x;
            size_t schedule_offset_barrir = blockDim.x * blockIdx.x;
            size_t vid = 0;
            while (__syncthreads_or(schedule_offset_barrir < vert_num)) {
                vid = schedule_offset_barrir + thread_idx;
                if (vid < vert_num) {
                    if (isactive[vid]) {
                        isactive[vid] = false;
                    }
                }
                __syncthreads();

                schedule_offset_barrir += blockDim.x * gridDim.x;
            }

            __syncthreads();
            __threadfence();
            if (threadIdx.x == 0) {
                atomicAdd((int *)&g_mutex2, 1);
                while (g_mutex2 < gridDim.x * iteration_id[l_iteration_id]) {}

            }
            __syncthreads();
        }
    }


}



