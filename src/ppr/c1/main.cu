#include "hip/hip_runtime.h"
#include "Util.cuh"
#include <time.h>
#include "deviceMemory.cuh"
#include <hip/hip_runtime.h>
#include <sys/time.h>

__device__ volatile int g_mutex1;
__device__ volatile int g_mutex2;
__device__ volatile int g_mutex3;
__device__ volatile int g_mutex4; // rule 遍历
__device__ volatile int g_mutex5; // rule 消息传播
__device__ volatile int g_mutex6; // rule 消息传播
__device__ volatile int g_mutex7; // rule 消息传播


template <typename ValueType>
__global__ void calcuatePPR(const int *csr_v, const int *csr_e, ValueType *pagerank,
            ValueType *residual, ValueType *messages, int *active_vert,
            int *active_verts_num, bool *isactive, int *degree, int *csr_o, int *csr_r,const int vert_num,
            const int rule_num, const ValueType alpha, const ValueType rmax,
            int source, int *iteration_id, int max_step,int *iteration_num,int *iteration_act_num,int *iter);
__global__ void printHello() {
    // 每个线程打印一次 "Hello"
    printf("Hello\n");
}

// Dump results
void DumpResults(const int verts_num, ValueType *d_pagerank,
                 ValueType *d_residual, ValueType *d_messages);

int main(int argc, char **argv) {
    // Initialize graph data in host & device memory
    hipFree(0);
    // 获取命令行参数
    std::string dir = argv[1];

    // 获取，csr_v ,csr_e ,v_r,degree,order;
    Graph graph(dir);
    if(graph.max_step<1) return 0;

    DeviceMemory device_memory(graph.vert_num, graph.rule_num, graph.edge_num,
                               graph.max_step);
    

    device_memory.CudaMemcpyGraph(graph);
    //std::cout << "test for study how to use cuda" << endl;

    hipStream_t stream;
    hipStreamCreate(&stream);

    int *iteration_id;
    hipMalloc(&iteration_id, sizeof(int) * 1000);
    int *iter;
    hipMalloc(&iter, sizeof(int) * (graph.max_step+2));
    int *iteration_num = new int[1];
	int *iteration_act_num = new int[1000];
	
	CUDA_ERROR(hipMemset(device_memory.active_verts_numStream, 0, sizeof(int)));//memset(指针， 初始值，大小）初始化
    // Initialize parameters for PPR
    int source = 0; // 101569
    ValueType alpha = 0.2f;
    ValueType rmax =0.01f * (1.0f / graph.origin_edge); // 这一步条件变了,应该是原始边数目

    struct timeval t_start, t_stop;
    double timeuse;
    gettimeofday(&t_start, NULL);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    //cout << "\n==================== PPR with FORWARD PUSH starts ""====================" << endl;
    //cout<<graph.vert_num<<"  "<<graph.rule_num<<endl;
    int cnt = 0;
    
     while (1) {
          if(graph.degree[source]==0){
			++source;
			continue;
		}
		
		cnt++;
		if(cnt%10==0){
		hipDeviceSynchronize();
		//cout << "source = " << source << "start ------------" <<cnt<< endl;

		}
         calcuatePPR<ValueType><<<MAX_BLOCKS_NUM, THREADS_PER_BLOCK>>>(
            device_memory.csr_v, device_memory.csr_e,
            device_memory.pagerankStream, device_memory.residualStream,
            device_memory.messagesStream, device_memory.active_vert,
            device_memory.active_verts_numStream,device_memory.isactive,device_memory.degree, 
            device_memory.csr_o, device_memory.csr_r,
            graph.vert_num,graph.rule_num, alpha, rmax, source++, iteration_id,
            graph.max_step,device_memory.iteration_num,device_memory.iteration_act_num,iter);
            if (cnt == 1) {
        	      hipDeviceSynchronize();
        	      //cout << "source = " << source << "end ------------" <<cnt<< endl;
                break;
            }
    }
    
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float runtime = 0; //milliseconds
	hipEventElapsedTime(&runtime, start, stop);
	cout << "gpu runtime: " << runtime/1000.0 << "： seconds" << endl;

	
	CUDA_ERROR(hipMemcpy(iteration_num, device_memory.iteration_num, sizeof(int)*1, hipMemcpyDeviceToHost));
	cout << "flag 已设置成 -1  终止条件以满足		iteration_num："<<iteration_num[0]<<endl;
	CUDA_ERROR(hipMemcpy(iteration_act_num, device_memory.iteration_act_num, sizeof(int)*1000, hipMemcpyDeviceToHost));
	
	cout << "0	:act_num: 1"<<endl;
	for(int i = 1 ;i<1000  ;i++){
		if(iteration_act_num[i]==0) break;
		cout <<i<< "	:act_num:	"<<iteration_act_num[i]<<endl;
		
	}
	cout << "==================== PPR with FORWARD PUSH ends ====================\n" << endl;

	//cout << "内存开辟耗时: " << timeMalloc << endl;
	gettimeofday(&t_stop, NULL);
	DumpResults(graph.vert_num, device_memory.pagerankStream, device_memory.residualStream, device_memory.messagesStream);

	timeuse = (t_stop.tv_sec - t_start.tv_sec) + (double)(t_stop.tv_usec - t_start.tv_usec)/1000000.0;
	//cout << "main total timeval runtime: " << timeuse << " seconds" << endl;
	return 0;

}

template <typename ValueType>
__global__ void calcuatePPR(const int *csr_v, const int *csr_e, ValueType *pagerank,
            ValueType *residual, ValueType *messages, int *active_vert,
            int *active_verts_num, bool *isactive, int *degree, int *csr_o, int *csr_r,const int vert_num,
            const int rule_num, const ValueType alpha, const ValueType rmax,
            int source, int *iteration_id, int max_step,int *iteration_num,int *iteration_act_num,int *iter) {

    size_t thread_id = threadIdx.x;
    size_t schedule_offset_init = blockDim.x * blockIdx.x;
    size_t vid = 0;
    while (schedule_offset_init < vert_num + rule_num) {
        vid = schedule_offset_init + thread_id;
        // in the last batch, some threads are idle
        if (vid < vert_num + rule_num) {
            pagerank[vid] = 0;
            residual[vid] = 0;
            messages[vid] = 0;
            isactive[vid] = false;
        }

        if (vid < 1000) {
            iteration_id[vid] = vid;
            
        }
        if(vid<max_step + 1){
        		iter[vid] = 0;
        }
        schedule_offset_init +=blockDim.x * gridDim.x; // 块线程数目*网格块数=网格线程数目
    }
    __syncthreads(); // 确保第一个初始化过程内的线程块同步


    // prepare for the 1st iteration
    size_t global_id = thread_id + blockDim.x * blockIdx.x;
    if (global_id == 0) { 
    // 每一个块中线程为0的id，source=1,第一个顶点的residual值初始化为1，
        residual[source] = 1;
        *active_verts_num = 1;
        active_vert[0] = source; // 当前迭代中的活跃顶点
        pagerank[source] += alpha * residual[source];
        g_mutex1 = 0;
        g_mutex2 = 0;
        g_mutex4 = 0;
        g_mutex5 = 0;
        g_mutex6 = 0;
        g_mutex7 = 0;
        //g_mutex3 = 0;
        //printf("source = %d 初始化完成\n", source);
    }

    __syncthreads(); //__syncthreads()只能在一个线程块内使用，不能用于不同线程块之间的同步。
    __threadfence(); // 线程屏障函数，它确保所有线程都在此之前的所有内存操作都已经完成。
    if (threadIdx.x == 0) {
        atomicAdd((int *)&g_mutex3, 1);
        while ((g_mutex3 == 0) || (g_mutex3 % gridDim.x)) {
        } // 用于等待其他线程块的同步的循环。
    }
    __syncthreads();

    int l_iteration_id = 0;
    int l_active_verts_num = *active_verts_num;
    int vaddr = vert_num + rule_num;

    size_t lane_id = thread_id % THREADS_PER_WARP; // warp内线程的id
    size_t warp_id =thread_id / THREADS_PER_WARP; // the i-th warp (from 0) 当前块内warp的id

    typedef hipcub::BlockScan<int, THREADS_PER_BLOCK> BlockScan;

    __shared__ typename BlockScan::TempStorage block_temp_storage;

    volatile __shared__ int comm[THREADS_PER_BLOCK / THREADS_PER_WARP][3];
    //[256/32][3]第一维是多少个warp，256/32=8，8个warp，

    volatile __shared__ ValueType commr[THREADS_PER_BLOCK / THREADS_PER_WARP]; // 每个warp对应一个值

    volatile __shared__ ValueType commd[THREADS_PER_BLOCK / THREADS_PER_WARP]; // 每个warp对应一个值

    volatile __shared__ int comm2[THREADS_PER_BLOCK]; // 一维数组大小256，int

    volatile __shared__ ValueType commd2[THREADS_PER_BLOCK]; // out-degree

    volatile __shared__ ValueType commr2[THREADS_PER_BLOCK];
    while (*active_verts_num > 0&&l_iteration_id<200) { //*active_verts_num > 0       l_iteration_id<1

        l_iteration_id += 1;
        // pushmessages 当前活跃顶点发消息

        __syncthreads();
        int total_active_verts_num = *active_verts_num;
        
        __syncthreads();
	        __threadfence();
        if (thread_id == 0) {
            atomicAdd((int *)&g_mutex1, 1);
            while (g_mutex1 < gridDim.x * iteration_id[l_iteration_id]) {}
            
        }
        	__threadfence();
        
        size_t idx = 0;
        int u = 0;
        int e_start, e_end;
        int v, du;
        ValueType ru, msg;
        
            // 顶点遍历
            size_t schedule_1 = blockDim.x * blockIdx.x; // 块索引
            while (schedule_1 < total_active_verts_num) {
                idx = schedule_1 + thread_id;
                if (idx < total_active_verts_num) {
                    u = active_vert[idx];
                    ru = residual[u];
                    du = degree[u];
                    residual[u] = 0;
                    e_start = csr_v[u]; // start offset of outgoing edges of "u"
                                        // in "col_ind"
                    e_end = csr_v[u + 1]; // end offset of outgoing edges of "u"
                                          // in "col_ind" (exclusive)
                    //补全0出度点的发送方向为源顶点
                    if(csr_v[u]==csr_v[u + 1]) {
					e_start = csr_v[source];
					e_end = csr_v[source + 1];
					du = degree[source];
					
                    }
                } else {
                    e_start = 0;
                    e_end = 0;
                }
                // while(1)
                while ( __syncthreads_or((e_end - e_start) >= THREADS_PER_BLOCK)) {
                    if ((e_end - e_start) >= THREADS_PER_BLOCK) {
                        comm[0][0] =
                            thread_id; // I (thread_id) want to process the
                                       // active vertex assigned to me.
                    }
                    __syncthreads(); // all threads in one block vote to
                                     // processing their own vertices

                    if (comm[0][0] == thread_id) {
                        comm[0][1] = e_start; // the vertx owned by me will be
                                     // processed in this <1>-while loop.
                        comm[0][2] = e_end;
                        commr[0] = ru; // ru是u的残差
                        commd[0] = du; // du是u的真实度

                        e_start = e_end; // avoid processing this vertex
                                         // repeatedly in <2>&<3>-while
                    }
                    __syncthreads(); // all threads are ready to process the
                                     // selected vertex

                    size_t push_st = comm[0][1] + thread_id; // process the "push_st"-th
                                                // outgoing edge at first.
                    size_t push_ed = comm[0][2];

                    // <1.1>-while: block-granularity-outgoing edges
                    while (__syncthreads_or(push_st < push_ed)) {
                        if (push_st < push_ed) {
                            v = csr_e[push_st]; // target vertex id
                            msg = ((1 - alpha) * commr[0]) / commd[0]; // outdeg of the selected s, not "u"
                            atomicAdd(messages + v, msg);
                            isactive[v] =true;
                            
                            
                        }
                        push_st +=THREADS_PER_BLOCK; // until all outgoing edges of
                                               // "u" have been processed
                    }
                } // until all source vertices with
                  // "todo_edges_num>=THREADS_PER_BLOCK" have been processed

                // while(2)
                while (__any_sync(FULL_MASK,(e_end - e_start) >= THREADS_PER_WARP)) {
                    if ((e_end - e_start) >= THREADS_PER_WARP) {
                        comm[warp_id][0] = lane_id; // threads in the "warp_id"-th warp try to
                                     // vote
                    }
                    if (comm[warp_id][0] == lane_id) {
                        comm[warp_id][1] = e_start; // vertex owned by the "lane_id"-th thread
                                     // in a warp is scheduled
                        comm[warp_id][2] = e_end;
                        commr[warp_id] = ru;
                        commd[warp_id] = du;
                        e_start = e_end; // avoid processing this vertex
                                         // repeatedly in <3>-while
                    }
                    size_t push_st = comm[warp_id][1] + lane_id; // process the "push_st"-th
                                                    // outgoing edge at first.
                    size_t push_ed = comm[warp_id][2];

                    // <2.1>-while: warp-granularity-outgoing edges
                    while (__any_sync(FULL_MASK, push_st < push_ed)) {
                        if (push_st < push_ed) {
                            v = csr_e[push_st];
                            msg = ((1 - alpha) * commr[warp_id]) /(commd[warp_id]);
                            atomicAdd(messages + v, msg);
                            isactive[v] =true;
                        }
                        push_st +=THREADS_PER_WARP; // until all outgoing edges of
                                              // "u" have been processed
                    }
                } // until all source vertices with
                  // "todo_edges_num>=THREADS_PER_WARP" have been processed

                // while(3) then, the out-degree of "u" is less than
                // THREADS_PER_WARP(32)
                int thread_count = e_end - e_start;
                int scatter = 0, total = 0;

                __syncthreads();
                BlockScan(block_temp_storage).ExclusiveSum(thread_count, scatter, total); //
                __syncthreads(); // there are "total" edges left in every block
                int progress = 0;

                while (progress < total) {
                    int remain = total - progress;
                    while (scatter < (progress + THREADS_PER_BLOCK) &&
                           (e_start < e_end)) {
                        comm2[scatter - progress] = e_start; // 存U有的外邻居
                        commd2[scatter - progress] = du;     //
                        commr2[scatter - progress] = ru;
                        scatter++;
                        e_start++;
                    }
                    __syncthreads();
                    int cur_batch_count = min(remain, (int)THREADS_PER_BLOCK);
                    // how many threads are required?
                    if (thread_id < cur_batch_count) {
                        v = csr_e[comm2[thread_id]]; //!!!
                        msg = ((1 - alpha) * commr2[thread_id]) / commd2[thread_id];
                        atomicAdd(messages + v, msg);
                        isactive[v] =true;
                    }
                    __syncthreads();
                    progress += THREADS_PER_BLOCK;
                }
                // schedule (blockDim.x * gridDim.x) active vertices per
                // <0>-while loop
                schedule_1 += blockDim.x * gridDim.x;
            }
///        

        // 规则遍历
        int local_iter = 0,total_rule =0,strat_r =0;
        while (local_iter < max_step ) {
        
            local_iter++;
            __syncthreads();
            
            // 同步
             strat_r = csr_o[local_iter-1];
            total_rule = csr_o[local_iter]-csr_o[local_iter-1];
            __syncthreads();
            __threadfence();
           if (thread_id == 0) {
            
                atomicAdd((int *)&g_mutex5, 1);
                while (g_mutex5 <gridDim.x *(iteration_id[l_iteration_id - 1] * max_step +local_iter)) {}
            }
            
			
            __syncthreads();
            

            
                // barrier 将新的活跃顶点加入边界
                volatile __shared__ size_t output_cta_offset;
                thread_id = threadIdx.x;
                size_t schedule_2 = blockDim.x * blockIdx.x;
                size_t vid = 0;

                while (__syncthreads_or(schedule_2 < total_rule)) {
                    // 这个函数返回的是一个布尔值，表示所有线程块中至少有一个线程满足条件。
                    vid = schedule_2 + thread_id;

                    int thread_cnt = 0;
                    if (vid < total_rule) {
                    
                        if (isactive[csr_r[vid + strat_r]]) {

                            //residual[csr_r[vid + strat_r]] =messages[csr_r[vid + strat_r]];
                            // 用残差值来传递消息
                            //messages[csr_r[vid + strat_r]] =0;
                            // 消息值置0，保证下一次迭代没问题
                            isactive[csr_r[vid + strat_r]] = false;
                            thread_cnt = 1;
                        }
                    }
                    int scatter = 0, total = 0;

                    __syncthreads();
                    BlockScan(block_temp_storage).ExclusiveSum(thread_cnt, scatter, total);
                    __syncthreads();
                    if (thread_id == 0) {
                        output_cta_offset = atomicAdd(&iter[local_iter-1], total);
                        // run per block
                    }
                    __syncthreads();
                    if (thread_cnt > 0) {
                        active_vert[output_cta_offset + scatter] = csr_r[vid + csr_o[local_iter-1]];
                        
                    }
                    schedule_2 += blockDim.x * gridDim.x;
                    // 用于更新 schedule_offset_barrir
                    // 变量的值，并且在每个线程块内都会执行。
                } // 更新规则结束

                __syncthreads();
                total_active_verts_num = iter[local_iter-1];
                __threadfence();

                if (thread_id == 0) {
                    atomicAdd((int *)&g_mutex4, 1);
                    while (g_mutex4 <gridDim.x * (iteration_id[l_iteration_id - 1] * max_step  + local_iter)) {}
                    
                    
                }
                
                __syncthreads();



            //*active_verts_num 已经更新完毕，
            // 下一步是消息传递，不需要边界检测

            	 thread_id = threadIdx.x;
                size_t schedule_3 = blockDim.x * blockIdx.x; // 块索引
				
                   size_t idxx = 0;
                    
        			int u = 0;
        			int e_start, e_end;
        			int v, du;
        			ValueType ru, msg;

                while (schedule_3 < iter[local_iter-1]) {
                    
                    idxx = schedule_3 + thread_id;
                    
                    // printf("uuuu%d\n",thread_id);
                    if (idxx < iter[local_iter-1]) {
                        u = active_vert[idxx];
                        ru = messages[u];
                        messages[u] = 0;
                        e_start = csr_v[u]; // start offset of outgoing edges of
                                            // "u" in "col_ind"
                        e_end = csr_v[u + 1]; // end offset of outgoing edges of
                                              // "u" in "col_ind" (exclusive)
                    } else {
                        e_start = 0;
                        e_end = 0;
                    }
                    // while(1)
                    while (__syncthreads_or((e_end - e_start) >=THREADS_PER_BLOCK)) {
                        if ((e_end - e_start) >= THREADS_PER_BLOCK) {
                            comm[0][0] =thread_id; // I (thread_id) want to process the
                                           // active vertex assigned to me.
                        }
                        __syncthreads(); // all threads in one block vote to
                                         // processing their own vertices

                        if (comm[0][0] == thread_id) {
                            comm[0][1] = e_start; // the vertx owned by me will be
                                         // processed in this <1>-while loop.
                            comm[0][2] = e_end;
                            commr[0] = ru; // ru是u的残差

                            e_start = e_end; // avoid processing this vertex
                                             // repeatedly in <2>&<3>-while
                        }
                        __syncthreads(); // all threads are ready to process the
                                         // selected vertex

                        size_t push_st =
                            comm[0][1] + thread_id; // process the "push_st"-th
                                                    // outgoing edge at first.
                        size_t push_ed = comm[0][2];

                        // <1.1>-while: block-granularity-outgoing edges
                        while (__syncthreads_or(push_st < push_ed)) {
                            if (push_st < push_ed) {
                                v = csr_e[push_st]; // target vertex id
                                //printf("zzzzzz%d \n", v);
                                msg = commr[0]; // outdeg of the selected s, not
                                                // "u"
                                atomicAdd(messages + v, msg);
                                isactive[v] =true;
                            }
                            push_st +=THREADS_PER_BLOCK; // until all outgoing edges
                                                   // of "u" have been processed
                        }
                    } // until all source vertices with
                      // "todo_edges_num>=THREADS_PER_BLOCK" have been processed

                    // while(2)
                    while (__any_sync(FULL_MASK, (e_end - e_start) >= THREADS_PER_WARP)) {
                        if ((e_end - e_start) >= THREADS_PER_WARP) {
                            comm[warp_id][0] =lane_id; // threads in the "warp_id"-th warp try
                                         // to vote
                        }
                        if (comm[warp_id][0] == lane_id) {
                            comm[warp_id][1] =  e_start; // vertex owned by the "lane_id"-th
                                         // thread in a warp is scheduled
                            comm[warp_id][2] = e_end;
                            commr[warp_id] = ru;
                            e_start = e_end; // avoid processing this vertex
                                             // repeatedly in <3>-while
                        }
                        size_t push_st = comm[warp_id][1] + lane_id; // process the "push_st"-th
                                                  // outgoing edge at first.
                        size_t push_ed = comm[warp_id][2];

                        // <2.1>-while: warp-granularity-outgoing edges
                        while (__any_sync(FULL_MASK, push_st < push_ed)) {
                            if (push_st < push_ed) {

                                v = csr_e[push_st];
                                // printf("xxxxxx%d \n",v);
                                msg = commr[warp_id];
                                atomicAdd(messages + v, msg);
                                isactive[v] =true;
                            }
                            push_st += THREADS_PER_WARP; // until all outgoing edges of
                                                  // "u" have been processed
                        }
                    } // until all source vertices with
                      // "todo_edges_num>=THREADS_PER_WARP" have been processed

                    // while(3) then, the out-degree of "u" is less than
                    // THREADS_PER_WARP(32)
                    int thread_count = e_end - e_start;
                    int scatter = 0, total = 0;

                    __syncthreads();
                    BlockScan(block_temp_storage)
                        .ExclusiveSum(thread_count, scatter, total); //
                    __syncthreads(); // there are "total" edges left in every
                                     // block
                    int progress = 0;

                    while (progress < total) {
                        int remain = total - progress;
                        while (scatter < (progress + THREADS_PER_BLOCK) &&(e_start < e_end)) {
                            comm2[scatter - progress] =
                                e_start; // 存U有的外邻居
                            commr2[scatter - progress] = ru;
                            scatter++;
                            e_start++;
                        }
                        __syncthreads();
                        int cur_batch_count =
                            min(remain, (int)THREADS_PER_BLOCK);
                        // how many threads are required?
                        if (thread_id < cur_batch_count) {
                            v = csr_e[comm2[thread_id]]; //!!!

                            msg = commr2[thread_id];
                            atomicAdd(messages + v, msg);
                            isactive[v] =true;
                        }
                        __syncthreads();
                        progress += THREADS_PER_BLOCK;
                    }
                    // schedule (blockDim.x * gridDim.x) active vertices per
                    // <0>-while loop
                    schedule_3 += blockDim.x * gridDim.x;
                } // 规则消息传播结束
            


        } // 规则遍历结束
///
__threadfence();
       if (thread_id == 0) {
            atomicAdd((int *)&g_mutex6, 1);
            while (g_mutex6 < gridDim.x * iteration_id[l_iteration_id]) {
            }
        }
        __syncthreads();

        // 边界检测
        __syncthreads();
        *active_verts_num = 0;
        size_t schedule_offset_rinit = blockDim.x * blockIdx.x;
     size_t rvid = 0;
    while (schedule_offset_rinit < max_step + 1) {
        rvid = schedule_offset_rinit + thread_id;
        // in the last batch, some threads are idle
        if(rvid<max_step + 1 ){
        iter[rvid] = 0;
        }

        schedule_offset_rinit +=blockDim.x * gridDim.x; 
        // 块线程数目 * 网格块数 = 网格线程数目
    }
    
        __threadfence();
        if (thread_id == 0) {
            atomicAdd((int *)&g_mutex7, 1);
            while (g_mutex7 < gridDim.x * iteration_id[l_iteration_id]) {
            }
        }
        __syncthreads();

        // barrier 将新的活跃顶点加入边界
        volatile __shared__ size_t output_cta_offset;
        size_t thread_id = threadIdx.x;
        schedule_1 = blockDim.x * blockIdx.x;
        size_t vid = 0;

        while (__syncthreads_or(schedule_1 < vert_num)) {
            // 这个函数返回的是一个布尔值，表示所有线程块中至少有一个线程满足条件。
            vid = schedule_1 + thread_id;
            int thread_cnt = 0;
            if (vid < vert_num) {
                if (isactive[vid]) {
                    residual[vid] += messages[vid];
                    messages[vid] = 0;
                    isactive[vid] = false;
                    if (residual[vid] / (degree[vid]) >= rmax) {
                        // 执行边界检测标准，符合条件将标志位设>置为1
                        pagerank[vid] += alpha * residual[vid];
                        thread_cnt = 1;
                    }
                }
            }
            int scatter = 0, total = 0;

            __syncthreads();
            BlockScan(block_temp_storage)
                .ExclusiveSum(thread_cnt, scatter, total);
            __syncthreads();
            if (thread_id == 0) {
                output_cta_offset =atomicAdd(active_verts_num, total); // run per block
            }
            __syncthreads();
            if (thread_cnt > 0) {
                active_vert[output_cta_offset + scatter] = vid;
            }
            schedule_1 += blockDim.x * gridDim.x;
            // 用于更新 schedule_offset_barrir
            // 变量的值，并且在每个线程块内都会执行。
        } // 边界检测结束

        __syncthreads();
        __threadfence();
        if (thread_id == 0) {
            atomicAdd((int *)&g_mutex2, 1);
            while (g_mutex2 < gridDim.x * iteration_id[l_iteration_id]) {}
            
        }
        	__threadfence();
        if (thread_id == 0) {
               iteration_num[0] = l_iteration_id;
			if(l_iteration_id<1000)	iteration_act_num[l_iteration_id]= *active_verts_num;
        }
        __syncthreads();
        //l_active_verts_num = *active_verts_num;
        __syncthreads();

    } // while (*active_verts_num != 0);
}



// Dump result
void DumpResults(const int verts_num, ValueType *d_pagerank,
                 ValueType *d_residual, ValueType *d_messages) {
    ValueType *h_pagerank = new ValueType[verts_num];
    ValueType *h_residual = new ValueType[verts_num];
    ValueType *h_messages = new ValueType[verts_num];

    CUDA_ERROR(hipMemcpy(h_pagerank, d_pagerank, verts_num * sizeof(ValueType),
                          hipMemcpyDeviceToHost));

    CUDA_ERROR(hipMemcpy(h_residual, d_residual, verts_num * sizeof(ValueType),
                          hipMemcpyDeviceToHost));

    CUDA_ERROR(hipMemcpy(h_messages, d_messages, verts_num * sizeof(ValueType),
                          hipMemcpyDeviceToHost));

    ofstream outfile("/home/lhb/cucode/PPR/Judge/iter_2.txt");

    for (int i = 0; i <= 10; i++) {
        cout << i << ".\tpagerank\t " << h_pagerank[i] << "\tresidual\t"
             << h_residual[i] << endl;
    }

    for (int i = 0; i < verts_num; i++) {
        outfile << i;
        outfile << " ";
        outfile << h_pagerank[i];
        outfile << " ";
        outfile << h_residual[i];
        outfile << "\n";
    }

    delete[] h_residual;
    delete[] h_pagerank;
    delete[] h_messages;
    h_residual = NULL;
    h_pagerank = NULL;
    h_messages = NULL;
}
