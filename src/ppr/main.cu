
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

// CUDA错误检查宏
#define CUDA_CHECK(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    } \
} while (0)

__global__ void waitKernel() {
    // do nothing
}

void waitTime(int seconds) {
    auto startTime = std::chrono::high_resolution_clock::now();
    auto targetTime = startTime + std::chrono::seconds(seconds);
    do {
        waitKernel<<<1, 1>>>();
        hipDeviceSynchronize();
    } while (std::chrono::high_resolution_clock::now() < targetTime);
}

int main() {
    // 申请6GB的显存
    int* degree;
    CUDA_CHECK(hipMalloc(&degree, sizeof(int) * 1024*1024*1024));

    // 等待一段时间（100秒）
    waitTime(10000);

    // 释放显存
    CUDA_CHECK(hipFree(degree));

    // 结束进程
    exit(EXIT_SUCCESS);
}