#include "hip/hip_runtime.h"
#include <time.h>
#include "Util.cuh"
#include "DeviceMemory.cuh"
#include "PPR.h"
#include <sys/time.h>

#include <hip/hip_runtime.h>
using namespace std;
__device__ volatile int g_mutex1;
__device__ volatile int g_mutex2;
__device__ volatile int g_mutex3;
__device__ volatile int g_mutex4; // rule 遍历
__device__ volatile int g_mutex5; // rule 消息传播
__device__ volatile int g_mutex6; // rule 消息传播
__device__ volatile int g_mutex7; // rule 消息传播

template <typename ValueType>
__global__ void
calcuatePPR(const int *csr_v, const int *csr_e, const ValueType *csr_w,ValueType *pagerank,
            ValueType *residual, ValueType *messages, int *active_vert,
            int *active_verts_num, bool *isactive, const int vert_num,
            const ValueType alpha, const ValueType rmax,
            int source, int *iteration_id,const ValueType *ww,int iter, int acc);

// Dump results
void DumpResults(const int verts_num, ValueType *d_pagerank, ValueType *d_residual, ValueType *d_messages);

int main(int argc, char **argv) {
    // Initialize graph data in host & device memory
    hipFree(0);
    // 获取命令行参数
    std::string dir = argv[1];
    int source = atoi(argv[2]);
    int iter = atoi(argv[3]);
  //  int acc = atoi(argv[4]);
    int acc = 100;
    int iterx=iter;
    graph.Graphinit(dir);

    DeviceMemory device_memory(graph.vert_num,  graph.edge_num);

    device_memory.CudaMemcpyGraph(graph);
    std::cout << "test for study how to use cuda" << endl;

    int *iteration_id;
    hipMalloc(&iteration_id, sizeof(int) * 2000);
    ValueType *wal;
    hipMalloc(&wal, sizeof(ValueType)*20000);
    int init_active_num1;
    int init_active_num2;


    // Initialize parameters for PPR


    //init bufferqueue
    BufferQueue *bufferqueue;
    bufferqueue = new BufferQueue;
    initQueue(bufferqueue, 20);

    //create thread
    pthread_t Pthread;

    if(0 != pthread_create(&Pthread, NULL, ppr_CPU, (void*)bufferqueue)){
        printf("Error:线程创建失败\n");
        exit(-1);
    }

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    cout << "\n==================== PPR with FORWARD PUSH starts ====================" << endl;


    // Initialize parameters for PPR

    int *flagG;
    CUDA_ERROR(hipMalloc(&flagG, sizeof(int)));
    CUDA_ERROR(hipMemset(flagG, -1, sizeof(int)));

    struct timeval t_start, t_stop;
    double timeuse;
    gettimeofday(&t_start, NULL);

    int cnt = 0;
    while (1) {

        //cout << "source = " << source << "start ------------" << endl;

        while(graph.csr_ov[source+1]-graph.csr_ov[source]<10||graph.indegree[source]<10){
            ++source;
        }
        init_active_num2 = graph.csr_ov[source+1]-graph.csr_ov[source];

        CUDA_ERROR(hipMemcpyAsync(device_memory.active_vert2, &graph.csr_oe[graph.csr_ov[source]],
                                   sizeof(int)*init_active_num2, hipMemcpyHostToDevice, stream2));

        CUDA_ERROR(hipMemcpyAsync(wal, &graph.csr_ow[graph.csr_ov[source]],
                                   sizeof(int)*init_active_num2, hipMemcpyHostToDevice, stream2));


        hipDeviceSynchronize();

        CUDA_ERROR(hipMemcpy(device_memory.active_vert2+init_active_num2, &graph.csr_e[graph.csr_v[source]],
                              sizeof(int)*(graph.csr_v[source+1]-graph.csr_v[source]), hipMemcpyHostToDevice));

        CUDA_ERROR(hipMemcpy(wal+init_active_num2, &graph.csr_w[graph.csr_v[source]],
                              sizeof(int)*(graph.csr_v[source+1]-graph.csr_v[source]), hipMemcpyHostToDevice));
        hipDeviceSynchronize();
        init_active_num2 += graph.csr_v[source+1]-graph.csr_v[source];
        hipDeviceSynchronize();


        CUDA_ERROR(hipMemcpyAsync(device_memory.active_verts_numStream2,&init_active_num2,
                                   sizeof(int), hipMemcpyHostToDevice, stream2));

        calcuatePPR<ValueType><<<MAX_BLOCKS_NUM, THREADS_PER_BLOCK, 0, stream2>>>(
                device_memory.csr_v, device_memory.csr_e,device_memory.csr_w,
                        device_memory.pagerankStream2, device_memory.residualStream2,
                        device_memory.messagesStream, device_memory.active_vert2,
                        device_memory.active_verts_numStream2,
                        device_memory.isactive, graph.vert_num,
                        alpha, graph.rmax, source++, iteration_id,wal,iter,acc);
        if(cnt){
            CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->global_ft_cnt, device_memory.active_verts_numStream1,
                                       sizeof(int), hipMemcpyDeviceToHost, stream1));

            CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->reserve, device_memory.pagerankStream1,
                                       graph.vert_num*sizeof(ValueType), hipMemcpyDeviceToHost, stream1));

            CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->residue, device_memory.residualStream1,
                                       graph.vert_num*sizeof(ValueType), hipMemcpyDeviceToHost, stream1));

            CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->global_ft, device_memory.active_vert1,
                                       graph.vert_num *sizeof(int), hipMemcpyDeviceToHost, stream1));

            //这个是关键同步机制？？？
            CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->flag, flagG,
                                       sizeof(int), hipMemcpyDeviceToHost, stream1));

            hipStreamSynchronize(stream1);
             if(bufferqueue->length>10){
                if(bufferqueue->length<15){
                    bufferqueue->front->source = source-1;
                    bufferqueue->front = bufferqueue->front->next; // 指针后移
                    bufferqueue->length++;
                }else{
                    iter=10000;
                }

            }else{
                bufferqueue->front->source = source-1;
                bufferqueue->front = bufferqueue->front->next; // 指针后移
                bufferqueue->length++;
                iter=iterx;
            }
            cnt++;
        }
        //hipDeviceSynchronize();
        while(graph.csr_ov[source+1]-graph.csr_ov[source]<10||graph.indegree[source]<10){
            ++source;
        }
        init_active_num1 = graph.csr_ov[source+1]-graph.csr_ov[source];

        CUDA_ERROR(hipMemcpyAsync(device_memory.active_vert1, &graph.csr_oe[graph.csr_ov[source]],
                                   sizeof(int)*init_active_num1, hipMemcpyHostToDevice, stream1));

        CUDA_ERROR(hipMemcpyAsync(wal, &graph.csr_ow[graph.csr_ov[source]],
                                   sizeof(int)*init_active_num1, hipMemcpyHostToDevice, stream1));

        hipDeviceSynchronize();

        CUDA_ERROR(hipMemcpy(device_memory.active_vert1+init_active_num1, &graph.csr_e[graph.csr_v[source]],
                              sizeof(int)*(graph.csr_v[source+1]-graph.csr_v[source]), hipMemcpyHostToDevice));

        CUDA_ERROR(hipMemcpy(wal+init_active_num1, &graph.csr_w[graph.csr_v[source]],
                              sizeof(int)*(graph.csr_v[source+1]-graph.csr_v[source]), hipMemcpyHostToDevice));
        hipDeviceSynchronize();
        init_active_num1 += graph.csr_v[source+1]-graph.csr_v[source];
        hipDeviceSynchronize();



        CUDA_ERROR(hipMemcpyAsync(device_memory.active_verts_numStream1,&init_active_num1,
                                   sizeof(int), hipMemcpyHostToDevice, stream1));

        calcuatePPR<ValueType><<<MAX_BLOCKS_NUM, THREADS_PER_BLOCK, 0, stream1>>>(
                device_memory.csr_v, device_memory.csr_e,device_memory.csr_w,
                        device_memory.pagerankStream1, device_memory.residualStream1,
                        device_memory.messagesStream, device_memory.active_vert1,
                        device_memory.active_verts_numStream1,
                        device_memory.isactive, graph.vert_num,
                        alpha, graph.rmax, source++, iteration_id,wal,iter,acc);



        //cout << "source = " << source - 1 << "end ------------" << endl;

        CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->global_ft_cnt, device_memory.active_verts_numStream2,
                                   sizeof(int), hipMemcpyDeviceToHost, stream2));

        CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->reserve, device_memory.pagerankStream2,
                                   graph.vert_num*sizeof(ValueType), hipMemcpyDeviceToHost, stream2));

        CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->residue, device_memory.residualStream2,
                                   graph.vert_num*sizeof(ValueType), hipMemcpyDeviceToHost, stream2));

        CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->global_ft, device_memory.active_vert2,
                                   graph.vert_num * sizeof(int), hipMemcpyDeviceToHost, stream2));

        CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->flag, flagG,
                                   sizeof(int), hipMemcpyDeviceToHost, stream2));


        // Stream2 完成后，更新队列
        hipStreamSynchronize(stream2);
         if(bufferqueue->length>10){
                if(bufferqueue->length<15){
                    bufferqueue->front->source = source-1;
                    bufferqueue->front = bufferqueue->front->next; // 指针后移
                    bufferqueue->length++;
                }else{
                    iter=10000;
                }

            }else{
                bufferqueue->front->source = source-1;
                bufferqueue->front = bufferqueue->front->next; // 指针后移
                bufferqueue->length++;
                iter=iterx;
            }
        cnt++;

        //hipDeviceSynchronize();

         cout << "当前bufferqueue长度为:\t" << bufferqueue->length << endl;
        if (cnt >=50) {
            CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->global_ft_cnt, device_memory.active_verts_numStream1,
                                       sizeof(int), hipMemcpyDeviceToHost, stream1));

            CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->reserve, device_memory.pagerankStream1,
                                       graph.vert_num*sizeof(ValueType), hipMemcpyDeviceToHost, stream1));

            CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->residue, device_memory.residualStream1,
                                       graph.vert_num*sizeof(ValueType), hipMemcpyDeviceToHost, stream1));

            CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->global_ft, device_memory.active_vert1,
                                       graph.vert_num *sizeof(int), hipMemcpyDeviceToHost, stream1));

            //这个是关键同步机制？？？
            CUDA_ERROR(hipMemcpyAsync(bufferqueue->front->flag, flagG,
                                       sizeof(int), hipMemcpyDeviceToHost, stream1));


            hipStreamSynchronize(stream1);
            bufferqueue->front->source = source-1;
            bufferqueue->front = bufferqueue->front->next; // 指针后移
            bufferqueue->length++;
            cnt++;
            hipDeviceSynchronize();
            bufferqueue->flag = -1;
            cout << "缓冲队列已空  图计算完成" << endl;


            hipEventRecord(stop);
            hipEventSynchronize(stop);
            float runtime = 0; //milliseconds
            hipEventElapsedTime(&runtime, start, stop);
            cout << "gpu runtime: " << runtime/1000.0 << " seconds" << endl;
            break;


        }
    }

    cout << "==================== PPR with FORWARD PUSH ends ====================\n" << endl;

    //cout << "内存开辟耗时: " << timeMalloc << endl;
    //DumpResults(graph.get_number_vertices(), device_memory.pagerank, device_memory.residual, device_memory.messages);
    pthread_join(Pthread, NULL);
//CPU执行之后的迭代过程

    gettimeofday(&t_stop, NULL);
    timeuse = (t_stop.tv_sec - t_start.tv_sec) + (double)(t_stop.tv_usec - t_start.tv_usec)/1000000.0;
    cout << "main total timeval runtime: " << timeuse << " seconds" << endl;
    return 0;
}


template <typename ValueType>
__global__ void
calcuatePPR(const int *csr_v, const int *csr_e, const ValueType *csr_w,ValueType *pagerank,
            ValueType *residual, ValueType *messages, int *active_vert,
            int *active_verts_num, bool *isactive, const int vert_num,
            const ValueType alpha, const ValueType rmax,
            int source, int *iteration_id,const ValueType *ww,int iter,int acc) {

    size_t thread_id = threadIdx.x;
    size_t schedule_offset_init = blockDim.x * blockIdx.x;
    size_t vid = 0;
    while (schedule_offset_init < vert_num) {
        vid = schedule_offset_init + thread_id;
        //in the last batch, some threads are idle
        if (vid < vert_num) {
            pagerank[vid] = 0;
            residual[vid] = 0;
            isactive[vid] = false;
            messages[vid] = 0;
        }
        if (vid < 2000) {
            iteration_id[vid] = vid;
        }
        schedule_offset_init += blockDim.x * gridDim.x;
    }

    //prepare for the 1st iteration
    size_t global_id = thread_id + blockDim.x*blockIdx.x;
    if (global_id == 0) { //每一个块中线程为0的id，source=1,第一个顶点的residual值初始化为1，
        residual[source] = 0;
        pagerank[source] = alpha;
        g_mutex1 = 0;
        g_mutex2 = 0;
        g_mutex4 = 0;
        g_mutex5 = 0;
        g_mutex6 = 0;

    }
    ////同步点

    __syncthreads();
    __threadfence();
    if (threadIdx.x == 0) {
        atomicAdd((int*) &g_mutex3, 1);
        while ((g_mutex3 == 0) || (g_mutex3 % gridDim.x) ) {}
        //	printf("g_mutex1 = %d \t阈值 : %d\n", g_mutex1, gridDim.x * iteration_id[l_iteration_id]);
    }
    __syncthreads();
    if(global_id<*active_verts_num){
        pagerank[active_vert[global_id]]+=(1-alpha)*alpha*ww[global_id];
        residual[active_vert[global_id]]+=(1-alpha)*ww[global_id];
    }



    __threadfence();
    if (threadIdx.x == 0) {
        atomicAdd((int*) &g_mutex6, 1);
        while ((g_mutex6 == 0) || (g_mutex6 % gridDim.x) ) {}
    }
    __syncthreads();

    int l_iteration_id = 0;
    int l_active_verts_num = *active_verts_num;

    size_t lane_id = thread_id % THREADS_PER_WARP; // warp内线程的id
    size_t warp_id =
            thread_id / THREADS_PER_WARP; // the i-th warp (from 0) 当前块内warp的id

    typedef hipcub::BlockScan<int, THREADS_PER_BLOCK> BlockScan;

    __shared__ typename BlockScan::TempStorage block_temp_storage;

    volatile __shared__ int comm[THREADS_PER_BLOCK / THREADS_PER_WARP][3];
    //[256/32][3]第一维是多少个warp，256/32=8，8个warp，

    volatile __shared__ ValueType
    commr[THREADS_PER_BLOCK / THREADS_PER_WARP]; // 每个warp对应一个值

    volatile __shared__ ValueType
    commd[THREADS_PER_BLOCK / THREADS_PER_WARP]; // 每个warp对应一个值

    volatile __shared__ int comm2[THREADS_PER_BLOCK]; // 一维数组大小256，int

    volatile __shared__ ValueType commd2[THREADS_PER_BLOCK]; // out-degree

    volatile __shared__ ValueType commr2[THREADS_PER_BLOCK];

    while ((*active_verts_num > 0)&&((l_iteration_id<iter)||(*active_verts_num>50000)) ){

        l_iteration_id += 1;
        // pushmessages 当前活跃顶点发消息

        __syncthreads();
        int total_active_verts_num = *active_verts_num;

        size_t idx = 0;
        int u = 0;
        int e_start, e_end;
        int v;
        ValueType ru, msg;

        // 顶点遍历
        size_t schedule_1 = blockDim.x * blockIdx.x; // 块索引
        while (schedule_1 < total_active_verts_num) {
            idx = schedule_1 + thread_id;
            if (idx < total_active_verts_num) {
                u = active_vert[idx];
                ru = residual[u];
                residual[u] = 0;
                e_start = csr_v[u]; // start offset of outgoing edges of "u"
                // in "col_ind"
                e_end = csr_v[u + 1]; // end offset of outgoing edges of "u"
                // in "col_ind" (exclusive)
            } else {
                e_start = 0;
                e_end = 0;
            }
            // while(1)
            while (
                    __syncthreads_or((e_end - e_start) >= THREADS_PER_BLOCK)) {
                if ((e_end - e_start) >= THREADS_PER_BLOCK) {
                    comm[0][0] =
                            thread_id; // I (thread_id) want to process the
                    // active vertex assigned to me.
                }
                __syncthreads(); // all threads in one block vote to
                // processing their own vertices

                if (comm[0][0] == thread_id) {
                    comm[0][1] =
                            e_start; // the vertx owned by me will be
                    // processed in this <1>-while loop.
                    comm[0][2] = e_end;
                    commr[0] = ru; // ru是u的残差

                    e_start = e_end; // avoid processing this vertex
                    // repeatedly in <2>&<3>-while
                }
                __syncthreads(); // all threads are ready to process the
                // selected vertex

                size_t push_st =
                        comm[0][1] + thread_id; // process the "push_st"-th
                // outgoing edge at first.
                size_t push_ed = comm[0][2];

                // <1.1>-while: block-granularity-outgoing edges
                while (__syncthreads_or(push_st < push_ed)) {
                    if (push_st < push_ed) {
                        v = csr_e[push_st]; // target vertex id

                        msg = ((1 - alpha) * commr[0]*csr_w[push_st]);// outdeg of the selected s, not "u"

                        atomicAdd(messages + v, msg);

                        isactive[v] =true;


                    }
                    push_st +=
                            THREADS_PER_BLOCK; // until all outgoing edges of
                    // "u" have been processed
                }
            } // until all source vertices with
            // "todo_edges_num>=THREADS_PER_BLOCK" have been processed

            // while(2)
            while (__any_sync(FULL_MASK,
                              (e_end - e_start) >= THREADS_PER_WARP)) {
                if ((e_end - e_start) >= THREADS_PER_WARP) {
                    comm[warp_id][0] =
                            lane_id; // threads in the "warp_id"-th warp try to
                    // vote
                }
                if (comm[warp_id][0] == lane_id) {
                    comm[warp_id][1] =
                            e_start; // vertex owned by the "lane_id"-th thread
                    // in a warp is scheduled
                    comm[warp_id][2] = e_end;
                    commr[warp_id] = ru;
                    e_start = e_end; // avoid processing this vertex
                    // repeatedly in <3>-while
                }
                size_t push_st =
                        comm[warp_id][1] + lane_id; // process the "push_st"-th
                // outgoing edge at first.
                size_t push_ed = comm[warp_id][2];

                // <2.1>-while: warp-granularity-outgoing edges
                while (__any_sync(FULL_MASK, push_st < push_ed)) {
                    if (push_st < push_ed) {
                        v = csr_e[push_st];
                        msg = ((1 - alpha) * commr[warp_id]*csr_w[push_st]);
                        atomicAdd(messages + v, msg);
                        isactive[v] =true;
                    }
                    push_st +=
                            THREADS_PER_WARP; // until all outgoing edges of
                    // "u" have been processed
                }
            } // until all source vertices with
            // "todo_edges_num>=THREADS_PER_WARP" have been processed

            // while(3) then, the out-degree of "u" is less than
            // THREADS_PER_WARP(32)
            int thread_count = e_end - e_start;
            int deg = thread_count;
            int scatter = 0, total = 0;

            __syncthreads();
            BlockScan(block_temp_storage)
                    .ExclusiveSum(thread_count, scatter, total); //
            __syncthreads(); // there are "total" edges left in every block
            int progress = 0;

            while (progress < total) {
                int remain = total - progress;
                while (scatter < (progress + THREADS_PER_BLOCK) &&
                       (e_start < e_end)) {
                    comm2[scatter - progress] = e_start; // 存U有的外邻居
                    commd2[scatter - progress] = deg;     //
                    commr2[scatter - progress] = ru;
                    scatter++;
                    e_start++;
                }
                __syncthreads();
                int cur_batch_count = min(remain, (int)THREADS_PER_BLOCK);
                // how many threads are required?
                if (thread_id < cur_batch_count) {

                    v = csr_e[comm2[thread_id]]; //!!!
                    //printf("the vert :%d \n",v);
                    msg = ((1 - alpha) * commr2[thread_id] * csr_w[comm2[thread_id]] ) ;
                    atomicAdd(messages + v, msg);
                    isactive[v] =true;
                }
                __syncthreads();
                progress += THREADS_PER_BLOCK;
            }
            // schedule (blockDim.x * gridDim.x) active vertices per
            // <0>-while loop
            schedule_1 += blockDim.x * gridDim.x;
        }
        //iter[0] = 0;



        // 边界检测
        __syncthreads();
        //host 有一个操作，将 active_verts_num 设置为0
        *active_verts_num = 0;

        __threadfence();
        if (threadIdx.x == 0) {
            atomicAdd((int*) &g_mutex1, 1);
            while (g_mutex1 < gridDim.x * iteration_id[l_iteration_id]) {}

        }
        __syncthreads();


        // barrier 将新的活跃顶点加入边界
        volatile __shared__ size_t output_cta_offset;
        size_t thread_idx = threadIdx.x;
        schedule_1 = blockDim.x * blockIdx.x;
        size_t vid = 0;

        while (__syncthreads_or(schedule_1 < vert_num)) {
            // 这个函数返回的是一个布尔值，表示所有线程块中至少有一个线程满足条件。
            vid = schedule_1 + thread_idx;
            int thread_cnt = 0;
            if (vid < vert_num) {
                if (isactive[vid]) {
                    residual[vid] += messages[vid];
                    messages[vid] = 0;
                    isactive[vid] = false;
                    if (residual[vid]  >(csr_v[vid+1]-csr_v[vid])* rmax) {
                        // 执行边界检测标准，符合条件将标志位设>置为1
                        pagerank[vid] += alpha * residual[vid];
                        thread_cnt = 1;
                    }
                }
            }
            int scatter = 0, total = 0;

            __syncthreads();
            BlockScan(block_temp_storage)
                    .ExclusiveSum(thread_cnt, scatter, total);
            __syncthreads();
            if (thread_id == 0) {
                output_cta_offset =
                        atomicAdd(active_verts_num, total); // run per block
            }
            __syncthreads();
            if (thread_cnt > 0) {
                active_vert[output_cta_offset + scatter] = vid;
            }
            schedule_1 += blockDim.x * gridDim.x;
            // 用于更新 schedule_offset_barrir
            // 变量的值，并且在每个线程块内都会执行。
        } // 边界检测结束

        __syncthreads();
        __threadfence();
        if (thread_id == 0) {
            atomicAdd((int *)&g_mutex2, 1);
            while (g_mutex2 < gridDim.x * iteration_id[l_iteration_id]) {
            }
        }
        __syncthreads();
        l_active_verts_num = *active_verts_num;

        // break;
    } // while (*active_verts_num != 0);
}


//Dump result
void DumpResults(const int verts_num, ValueType *d_pagerank, ValueType *d_residual, ValueType *d_messages) {
    ValueType *h_pagerank = new ValueType[verts_num];
    ValueType *h_residual = new ValueType[verts_num];
    ValueType *h_messages = new ValueType[verts_num];

    CUDA_ERROR(hipMemcpy(h_pagerank, d_pagerank,
                          verts_num*sizeof(ValueType), hipMemcpyDeviceToHost));

    CUDA_ERROR(hipMemcpy(h_residual, d_residual,
                          verts_num*sizeof(ValueType), hipMemcpyDeviceToHost));

    CUDA_ERROR(hipMemcpy(h_messages, d_messages,
                          verts_num*sizeof(ValueType), hipMemcpyDeviceToHost));

    ofstream outfile("/home/lhb/cucode/PPR/Judge/iter_2.txt");

    for (int i = 0; i <=10; i++) {
        cout<<i<<".\tpageran\t "<<h_pagerank[i] << "\tresidual\t" <<h_residual[i] <<endl;
    }

    for (int i = 0; i < verts_num; i++) {
        outfile<<i;
        outfile<<" ";
        outfile<<h_pagerank[i];
        outfile<<" ";
        outfile<<h_residual[i];
        outfile<<"\n";
    }

    delete[] h_residual;
    delete[] h_pagerank;
    delete[] h_messages;
    h_residual = NULL;
    h_pagerank = NULL;
    h_messages = NULL;
}

