#include "hip/hip_runtime.h"
#include <time.h>
#include "Util.cuh"
#include "DeviceMemory.cuh"

__device__ volatile int g_mutex1;
__device__ volatile int g_mutex2;
__device__ volatile int g_mutex3;
__device__ volatile int g_mutex4;
__device__ volatile int g_mutex5;
__device__ volatile int g_mutex6;
//原子操作重写
__device__ static float atomicMin(float *address, float val) {
    int *address_as_i = (int *) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
                          __float_as_int(::fminf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

//kernel_ update
template<typename ValueType>
__global__ void CalcuPPR(const Vertex *csr_v, const Vertex *csr_e, const ValueType *csr_w,
                         ValueType *pagerank, ValueType *residual, ValueType *messages,
                          const Vertex *csr_ov, const Vertex *csr_idx,
                          Vertex *active_vert1,Vertex *active_vert2, Vertex *active_vert,ValueType *active_w1,ValueType *active_w2,
                          Vertex *active_vert_num1,Vertex *active_vert_num2,Vertex *active_vert_num, bool *isactive,
                          const Vertex vert_num, Vertex source, Vertex *iteration_id, Vertex *iteration_num,
                          Vertex *iteration_act_num,ValueType alpha,ValueType rmax);

int main(int argc, char **argv) {
    // Initialize graph data in host & device memory
    hipFree(0);
    // 获取命令行参数
    std::string dir = argv[1];
    int source = atoi(argv[2]);

    // 获取，csr_v ,csr_e ,v_r,degree,order;
    Graph graph(dir);

    DeviceMemory device_memory(graph.vert_num, graph.rule_num,graph.edge_num,graph.w_num);

    device_memory.CudaMemcpyGraph(graph);
    std::cout << "test for study how to use cuda" << endl;
    int vert_num = graph.vert_num;
    int add_num = graph.add_num;
    int rule_num = graph.rule_num;
    int edge_num = graph.edge_num;


    int *iteration_id;
    iteration_id = NULL;
    hipMalloc(&iteration_id, sizeof(int) * 2000);

    int *iteration_num = new int[1];
    int *ac1 = new int[1];
    int *ac2 = new int[1];
    int *iteration_act_num = new int[2000];
    ValueType alpha = 0.2f;
    ValueType rmax =0.001f/(graph.vert_num);
    cout << "\n==================== SSSP with FORWARD PUSH starts ============" << endl;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);


    int cnt = 0;

    while (1) {
hipDeviceSynchronize();
        CalcuPPR<ValueType><<<MAX_BLOCKS_NUM, THREADS_PER_BLOCK>>>(
                    device_memory.csr_v, device_memory.csr_e, device_memory.csr_w,
                    device_memory.pagerankStream,device_memory.residualStream,device_memory.messagesStream,
                    device_memory.csr_ov, device_memory.csr_idx,
                    device_memory.active_vert1,device_memory.active_vert2,device_memory.active_vert,
                    device_memory.active_w1,device_memory.active_w2,
                    device_memory.active_vert_num1,device_memory.active_vert_num2, device_memory.active_vert_num,
                    device_memory.isactive,
                    vert_num, source, iteration_id, device_memory.iteration_num, device_memory.iteration_act_num, alpha, rmax);


        cnt++;
        if(cnt % 10 ==0){
            hipDeviceSynchronize();
        }
        if (cnt == 10) {
        	hipDeviceSynchronize();
            break;
        }

    }


    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    CUDA_ERROR(hipMemcpy(iteration_num, device_memory.iteration_num, sizeof(int) * 1, hipMemcpyDeviceToHost));
    
    cout << "flag 已设置成 -1  终止条件以满足		iteration_num：" << iteration_num[0] << endl;
    CUDA_ERROR(hipMemcpy(iteration_act_num, device_memory.iteration_act_num,sizeof(int) * 200, hipMemcpyDeviceToHost));
    cout << "0	act_num：1" << endl;
    for (int i = 1; iteration_act_num[i] != 0; i++) {
        cout << i << "	act_num：" << iteration_act_num[i] << endl;
        if (i > 198) break;
    }
    float runtime = 0;
    hipEventElapsedTime(&runtime, start, stop);
    cout << "gpu runtime: " << runtime / 1000.0 << " seconds" << endl;
    cout << "源顶点source = " << source - 1 << endl;

    return 0;
}


template<typename ValueType>
__global__ void CalcuPPR(const Vertex *csr_v, const Vertex *csr_e, const ValueType *csr_w,
                         ValueType *pagerank, ValueType *residual, ValueType *messages,
                         const Vertex *csr_ov, const Vertex *csr_idx,
                         Vertex *active_vert1,Vertex *active_vert2, Vertex *active_vert,ValueType *active_w1,ValueType *active_w2,
                         Vertex *active_vert_num1,Vertex *active_vert_num2,Vertex *active_vert_num, bool *isactive,
                         const Vertex vert_num, Vertex source, Vertex *iteration_id, Vertex *iteration_num,
                         Vertex *iteration_act_num,ValueType alpha,ValueType rmax) {
    size_t thread_id = threadIdx.x;
    size_t schedule_offset_init = blockDim.x * blockIdx.x;
    size_t vid = 0;

    while (schedule_offset_init < vert_num) {

        vid = schedule_offset_init + thread_id;

        if (vid < vert_num) {
            pagerank[vid] = 0;
            residual[vid] = 0;
            isactive[vid] = false;
            messages[vid] = 0;
        }
        if (vid < 2000) {
            iteration_id[vid] = vid;
        }
        schedule_offset_init += blockDim.x * gridDim.x;
    }

    //prepare for iteration
    size_t global_id = thread_id + blockDim.x * blockIdx.x;
    if (global_id == 0) {
        active_vert[0] = source;
        pagerank[source] = alpha * residual[source];
        residual[source] = 1;
        *active_vert_num1 = 0;
        *active_vert_num2 = 0;
        *active_vert_num = 1;
        g_mutex1 = 0;
        g_mutex2 = 0;
        g_mutex4 = 0;
        g_mutex5 = 0;
        g_mutex6 = 0;
    }

    __threadfence();
    if (threadIdx.x == 0) {
        atomicAdd((int*) &g_mutex3, 1);
        while ((g_mutex3 == 0) || (g_mutex3 % gridDim.x) ) {}
    }
    __syncthreads();
    int l_iteration_id = 0;
    int total_avtive_num;
    thread_id = threadIdx.x;//当前块内的线程id
    size_t lane_id = thread_id % THREADS_PER_WARP;
    size_t warp_id = thread_id / THREADS_PER_WARP;

    typedef hipcub::BlockScan<int, THREADS_PER_BLOCK> BlockScan;
    __shared__ typename BlockScan::TempStorage block_temp_storage;

    volatile __shared__ int comm[THREADS_PER_BLOCK / THREADS_PER_WARP][3];//[256/32][3]第一维是多少个warp，256/32=8，8个warp，
    volatile __shared__ ValueType commr[THREADS_PER_BLOCK / THREADS_PER_WARP];//每个warp对应一个值
    volatile __shared__ int commi[THREADS_PER_BLOCK / THREADS_PER_WARP][2];//每个warp对应3个值
    volatile __shared__ int comm2[THREADS_PER_BLOCK]; //一维数组大小256，int
    volatile __shared__ int commd2[THREADS_PER_BLOCK]; //out-degree
    volatile __shared__ ValueType commr2[THREADS_PER_BLOCK];
    volatile __shared__ int commi2[THREADS_PER_BLOCK][2];//每个warp对应3个值

    while ((*active_vert_num1 >0||*active_vert_num2 >0||*active_vert_num>0)&&l_iteration_id<2000) {
        l_iteration_id += 1;
        __syncthreads();
        __threadfence();
        if (threadIdx.x == 0) {
            atomicAdd((int *) &g_mutex6, 1);
            while (g_mutex6 < gridDim.x * iteration_id[l_iteration_id]) {}
            iteration_num[0] = l_iteration_id;
            if(l_iteration_id<2000)	iteration_act_num[l_iteration_id]= (*active_vert_num)+(*active_vert_num1)/2+(*active_vert_num2)/2;
        }
        __syncthreads();
        if(l_iteration_id%2==1){
            //先顶点遍历
            __syncthreads();
            total_avtive_num=*active_vert_num;

            __syncthreads();

            size_t schedule_offset = blockDim.x * blockIdx.x;
            size_t idx=0;
            int row_start=0, row_end=0,testx=0;
            int u=0, v=0, root=0, idxx=0;
            ValueType ru=0.0, msg=0.0; //dis value of u
            while (schedule_offset < total_avtive_num) {
                idx = schedule_offset + thread_id;
                if (idx < total_avtive_num) {
                    u = active_vert[idx];
                    //printf("%d",u);
                    ru =(1-alpha)* residual[u];
                    idxx=csr_ov[u];
                    residual[u] = 0;
                    row_start = csr_v[u]; //start offset of outgoing edges of "u" in "col_ind"
                    row_end = csr_v[u+1]; //end offset of outgoing edges of "u" in "col_ind" (exclusive)
                } else {
                    row_start = 0;
                    row_end = 0;
                }
                //while(1)
                while (__syncthreads_or((row_end-row_start)>=THREADS_PER_BLOCK)) {
                    if ((row_end-row_start) >= THREADS_PER_BLOCK) {
                        comm[0][0] = thread_id; //I (thread_id) want to process the active vertex assigned to me.
                    }
                    __syncthreads(); //all threads in one block vote to processing their own vertices

                    if (comm[0][0] == thread_id) {
                        comm[0][1] = row_start; //the vertx owned by me will be processed in this <1>-while loop.
                        comm[0][2] = row_end;
                        commr[0] = ru;//ru是u的残差
                        commi[0][1] = idxx;
                        row_start = row_end; //avoid processing this vertex repeatedly in <2>&<3>-while
                    }
                    __syncthreads(); //all threads are ready to process the selected vertex

                    size_t push_st = comm[0][1] + thread_id; //process the "push_st"-th outgoing edge at first.
                    size_t push_ed = comm[0][2];

                    // <1.1>-while: block-granularity-outgoing edges
                    while (__syncthreads_or(push_st<push_ed)) {
                        if (push_st < push_ed) {
                            v = csr_e[push_st]; //target vertex id
                            testx = commi[0][1] + csr_idx[push_st];
                            msg = commr[0]*csr_w[testx];
                            if(v<vert_num){
                                atomicAdd(messages + v, msg);
                                isactive[v] = true;
                            }else{
                                int cur = atomicAdd(active_vert_num1,2);
                                active_vert1[cur]=v;
                                active_vert1[cur+1]=testx;
                                active_w1[cur/2] = commr[0];
                            }
                        }
                        push_st += THREADS_PER_BLOCK; //until all outgoing edges of "u" have been processed
                    }
                } //until all source vertices with "todo_edges_num>=THREADS_PER_BLOCK" have been processed
                //while(2)
                while (__any_sync(FULL_MASK, (row_end-row_start)>=THREADS_PER_WARP)) {
                    if ((row_end-row_start) >= THREADS_PER_WARP) {
                        comm[warp_id][0] = lane_id; //threads in the "warp_id"-th warp try to vote
                    }
                    if (comm[warp_id][0] == lane_id) {
                        comm[warp_id][1] = row_start; //vertex owned by the "lane_id"-th thread in a warp is scheduled
                        comm[warp_id][2] = row_end;
                        commr[warp_id] = ru;
                        commi[warp_id][1] = idxx;
                        row_start = row_end; //avoid processing this vertex repeatedly in <3>-while
                    }
                    size_t push_st = comm[warp_id][1] + lane_id; //process the "push_st"-th outgoing edge at first.
                    size_t push_ed = comm[warp_id][2];

                    // <2.1>-while: warp-granularity-outgoing edges
                    while (__any_sync(FULL_MASK, push_st<push_ed)) {
                        if (push_st < push_ed) {
                            v = csr_e[push_st];
                            testx = commi[warp_id][1] + csr_idx[push_st];
                            msg = commr[warp_id]*csr_w[testx];
                            if(v<vert_num){
                                atomicAdd(messages + v, msg);
                                isactive[v] = true;
                            }else{
                                int cur = atomicAdd(active_vert_num1,2);
                                active_vert1[cur]=v;
                                active_vert1[cur+1]=testx;
                                active_w1[cur/2] = commr[warp_id];
                            }
                        }
                        push_st += THREADS_PER_WARP; //until all outgoing edges of "u" have been processed
                    }
                } //until all source vertices with "todo_edges_num>=THREADS_PER_WARP" have been processed

                //while(3) then, the out-degree of "u" is less than THREADS_PER_WARP(32)
                int thread_count = row_end - row_start;
                int deg = thread_count;
                int scatter = 0, total = 0;

                __syncthreads();
                BlockScan(block_temp_storage).ExclusiveSum(thread_count, scatter, total); //
                __syncthreads(); //there are "total" edges left in every block
                int progress = 0;
                while (progress < total) {
                    int remain = total - progress;
                    while (scatter<(progress+THREADS_PER_BLOCK) && (row_start<row_end)) {
                        comm2[scatter-progress] = row_start;//存U有的外邻居
                        commd2[scatter-progress] = deg; //
                        commr2[scatter-progress] = ru;
                        commi2[scatter - progress][1] = idxx;
                        scatter++;
                        row_start++;
                    }
                    __syncthreads();
                    int cur_batch_count = min(remain, (int)THREADS_PER_BLOCK); //how many threads are required?
                    if (thread_id < cur_batch_count) {
                        v = csr_e[comm2[thread_id]];
                        testx = commi2[thread_id][1] + csr_idx[comm2[thread_id]];
                        msg = (commr2[thread_id]) *csr_w[testx];
                        if(v<vert_num){
                            atomicAdd(messages + v, msg);
                            isactive[v] = true;
                        }else{
                            int cur = atomicAdd(active_vert_num1,2);
                            active_vert1[cur]=v;
                            active_vert1[cur+1]=testx;
                            active_w1[cur/2] =commr2[thread_id];
                        }
                    }
                    __syncthreads();
                    progress += THREADS_PER_BLOCK;
                }
                //schedule (blockDim.x * gridDim.x) active vertices per <0>-while loop
                schedule_offset += blockDim.x * gridDim.x;
            }
            __syncthreads();
            __threadfence();
            if (threadIdx.x == 0) {
                atomicAdd((int *) &g_mutex1, 1);
                while (g_mutex1 < gridDim.x * iteration_id[l_iteration_id]) {}
            }
            __syncthreads();
            //host 有一个操作，将 active_verts_num 设置为0
            *active_vert_num = 0;
            total_avtive_num= *active_vert_num1/2;
            __syncthreads();
            __threadfence();


            schedule_offset = blockDim.x * blockIdx.x;
            while (schedule_offset < total_avtive_num) {
                idx = schedule_offset + thread_id;
                if (idx < total_avtive_num) {
                    u = active_vert1[idx*2];
                    idxx=active_vert1[idx*2+1];
                    ru = active_w1[idx];
                    row_start = csr_v[u]; //start offset of outgoing edges of "u" in "col_ind"
                    row_end = csr_v[u+1]; //end offset of outgoing edges of "u" in "col_ind" (exclusive)
                } else {
                    row_start = 0;
                    row_end = 0;
                }
                //while(1)
                while (__syncthreads_or((row_end-row_start)>=THREADS_PER_BLOCK)) {
                    if ((row_end-row_start) >= THREADS_PER_BLOCK) {
                        comm[0][0] = thread_id; //I (thread_id) want to process the active vertex assigned to me.
                    }
                    __syncthreads(); //all threads in one block vote to processing their own vertices

                    if (comm[0][0] == thread_id) {
                        comm[0][1] = row_start; //the vertx owned by me will be processed in this <1>-while loop.
                        comm[0][2] = row_end;
                        commr[0] = ru;//ru是u的残差
                        commi[0][1] = idxx;
                        row_start = row_end; //avoid processing this vertex repeatedly in <2>&<3>-while
                    }
                    __syncthreads(); //all threads are ready to process the selected vertex

                    size_t push_st = comm[0][1] + thread_id; //process the "push_st"-th outgoing edge at first.
                    size_t push_ed = comm[0][2];

                    // <1.1>-while: block-granularity-outgoing edges
                    while (__syncthreads_or(push_st<push_ed)) {
                        if (push_st < push_ed) {
                            v = csr_e[push_st]; //target vertex id
                            testx = commi[0][1] + csr_idx[push_st];
                            msg = commr[0]*csr_w[testx];
                            if(v<vert_num){
                                atomicAdd(messages + v, msg);
                                isactive[v] = true;
                            }else{
                                int cur = atomicAdd(active_vert_num2,2);
                                active_vert2[cur]=v;
                                active_vert2[cur+1]=testx;
                                active_w2[cur/2] = commr[0];
                            }
                        }
                        push_st += THREADS_PER_BLOCK; //until all outgoing edges of "u" have been processed
                    }
                } //until all source vertices with "todo_edges_num>=THREADS_PER_BLOCK" have been processed
                //while(2)
                while (__any_sync(FULL_MASK, (row_end-row_start)>=THREADS_PER_WARP)) {
                    if ((row_end-row_start) >= THREADS_PER_WARP) {
                        comm[warp_id][0] = lane_id; //threads in the "warp_id"-th warp try to vote
                    }
                    if (comm[warp_id][0] == lane_id) {
                        comm[warp_id][1] = row_start; //vertex owned by the "lane_id"-th thread in a warp is scheduled
                        comm[warp_id][2] = row_end;
                        commr[warp_id] = ru;
                        commi[warp_id][1] = idxx;
                        row_start = row_end; //avoid processing this vertex repeatedly in <3>-while
                    }
                    size_t push_st = comm[warp_id][1] + lane_id; //process the "push_st"-th outgoing edge at first.
                    size_t push_ed = comm[warp_id][2];

                    // <2.1>-while: warp-granularity-outgoing edges
                    while (__any_sync(FULL_MASK, push_st<push_ed)) {
                        if (push_st < push_ed) {
                            v = csr_e[push_st];
                            testx = commi[warp_id][1] + csr_idx[push_st];
                            msg = commr[warp_id]*csr_w[testx];
                            if(v<vert_num){
                                atomicAdd(messages + v, msg);
                                isactive[v] = true;
                            }else{
                                int cur = atomicAdd(active_vert_num2,2);
                                active_vert2[cur]=v;
                                active_vert2[cur+1]=testx;
                                active_w2[cur/2] = commr[warp_id];
                            }
                        }
                        push_st += THREADS_PER_WARP; //until all outgoing edges of "u" have been processed
                    }
                } //until all source vertices with "todo_edges_num>=THREADS_PER_WARP" have been processed

                //while(3) then, the out-degree of "u" is less than THREADS_PER_WARP(32)
                int thread_count = row_end - row_start;
                int deg = thread_count;
                int scatter = 0, total = 0;

                __syncthreads();
                BlockScan(block_temp_storage).ExclusiveSum(thread_count, scatter, total); //
                __syncthreads(); //there are "total" edges left in every block
                int progress = 0;
                while (progress < total) {
                    int remain = total - progress;
                    while (scatter<(progress+THREADS_PER_BLOCK) && (row_start<row_end)) {
                        comm2[scatter-progress] = row_start;//存U有的外邻居
                        commd2[scatter-progress] = deg; //
                        commr2[scatter-progress] = ru;
                        commi2[scatter - progress][1] = idxx;
                        scatter++;
                        row_start++;
                    }
                    __syncthreads();
                    int cur_batch_count = min(remain, (int)THREADS_PER_BLOCK); //how many threads are required?
                    if (thread_id < cur_batch_count) {
                        v = csr_e[comm2[thread_id]];
                        testx = commi2[thread_id][1] + csr_idx[comm2[thread_id]];
                        msg = (commr2[thread_id]) *csr_w[testx];
                        if(v<vert_num){
                            atomicAdd(messages + v, msg);
                            isactive[v] = true;
                        }else{
                            int cur = atomicAdd(active_vert_num2,2);
                            active_vert2[cur]=v;
                            active_vert2[cur+1]=testx;
                            active_w2[cur/2] =commr2[thread_id];
                        }
                    }
                    __syncthreads();
                    progress += THREADS_PER_BLOCK;
                }
                //schedule (blockDim.x * gridDim.x) active vertices per <0>-while loop
                schedule_offset += blockDim.x * gridDim.x;
            }
            __syncthreads();



            __threadfence();
            if (threadIdx.x == 0) {
                atomicAdd((int*) &g_mutex2, 1);
                while (g_mutex2 < gridDim.x * iteration_id[l_iteration_id]) {}

            }
            __syncthreads();

            //barrier 将新的活跃顶点加入边界
            volatile __shared__ size_t output_cta_offset;
            size_t thread_idx = threadIdx.x;
            size_t schedule_offset_barrir = blockDim.x * blockIdx.x;
            while (__syncthreads_or(schedule_offset_barrir < vert_num)) {
                vid = schedule_offset_barrir + thread_idx;
                int thread_cnt = 0;
                if (vid < vert_num) {
                    if (isactive[vid]) {
                        residual[vid] += messages[vid];
                        messages[vid] = 0;
                        isactive[vid] = false;
                        if ((residual[vid]) > (csr_ov[vid+1] - csr_ov[vid])*rmax) {//执行边界检测标准，符合条件将标志位设>置为1
                            pagerank[vid] += alpha * residual[vid];
                            thread_cnt = 1;
                        }
                    }
                }
                int scatter = 0, total = 0;

                __syncthreads();
                BlockScan(block_temp_storage).ExclusiveSum(thread_cnt, scatter, total);
                __syncthreads();
                if (thread_id == 0) {
                    output_cta_offset = atomicAdd(active_vert_num, total); //run per block
                }
                __syncthreads();
                if (thread_cnt > 0) {
                    active_vert[output_cta_offset+scatter] = vid;
                }
                schedule_offset_barrir += blockDim.x * gridDim.x;
            }

            __syncthreads();
            __threadfence();
            if (thread_id == 0) {
                atomicAdd((int*) &g_mutex4, 1);
                while (g_mutex4 < gridDim.x * iteration_id[l_iteration_id]) {}

            }
            __syncthreads();

            *active_vert_num1 = 0;

            __threadfence();
            if (threadIdx.x == 0) {
                atomicAdd((int *) &g_mutex5, 1);
                while (g_mutex5 < gridDim.x * iteration_id[l_iteration_id]) {}
             //   iteration_num[0] = l_iteration_id;
              //  if(l_iteration_id<2000)	iteration_act_num[l_iteration_id]= (*active_vert_num)+(*active_vert_num2)/2;
                //printf("%d -",iteration_num[0]);
            }

            __threadfence();
        }else{
            //先顶点遍历
            __syncthreads();
            total_avtive_num=*active_vert_num;

            __syncthreads();

            size_t schedule_offset = blockDim.x * blockIdx.x;
            size_t idx=0;
            int row_start=0, row_end=0,testx=0;
            int u=0, v=0, root=0, idxx=0;
            ValueType ru=0.0, msg=0.0; //dis value of u
            while (schedule_offset < total_avtive_num) {
                idx = schedule_offset + thread_id;
                if (idx < total_avtive_num) {
                    u = active_vert[idx];
                    ru =(1-alpha)* residual[u];
                    idxx=csr_ov[u];
                    residual[u] = 0;
                    row_start = csr_v[u]; //start offset of outgoing edges of "u" in "col_ind"
                    row_end = csr_v[u+1]; //end offset of outgoing edges of "u" in "col_ind" (exclusive)
                } else {
                    row_start = 0;
                    row_end = 0;
                }
                //while(1)
                while (__syncthreads_or((row_end-row_start)>=THREADS_PER_BLOCK)) {
                    if ((row_end-row_start) >= THREADS_PER_BLOCK) {
                        comm[0][0] = thread_id; //I (thread_id) want to process the active vertex assigned to me.
                    }
                    __syncthreads(); //all threads in one block vote to processing their own vertices

                    if (comm[0][0] == thread_id) {
                        comm[0][1] = row_start; //the vertx owned by me will be processed in this <1>-while loop.
                        comm[0][2] = row_end;
                        commr[0] = ru;//ru是u的残差
                        commi[0][1] = idxx;
                        row_start = row_end; //avoid processing this vertex repeatedly in <2>&<3>-while
                    }
                    __syncthreads(); //all threads are ready to process the selected vertex

                    size_t push_st = comm[0][1] + thread_id; //process the "push_st"-th outgoing edge at first.
                    size_t push_ed = comm[0][2];

                    // <1.1>-while: block-granularity-outgoing edges
                    while (__syncthreads_or(push_st<push_ed)) {
                        if (push_st < push_ed) {
                            v = csr_e[push_st]; //target vertex id
                            testx = commi[0][1] + csr_idx[push_st];
                            msg = commr[0]*csr_w[testx];
                            if(v<vert_num){
                                atomicAdd(messages + v, msg);
                                isactive[v] = true;
                            }else{
                                int cur = atomicAdd(active_vert_num2,2);
                                active_vert2[cur]=v;
                                active_vert2[cur+1]=testx;
                                active_w2[cur/2] = commr[0];
                            }
                        }
                        push_st += THREADS_PER_BLOCK; //until all outgoing edges of "u" have been processed
                    }
                } //until all source vertices with "todo_edges_num>=THREADS_PER_BLOCK" have been processed
                //while(2)
                while (__any_sync(FULL_MASK, (row_end-row_start)>=THREADS_PER_WARP)) {
                    if ((row_end-row_start) >= THREADS_PER_WARP) {
                        comm[warp_id][0] = lane_id; //threads in the "warp_id"-th warp try to vote
                    }
                    if (comm[warp_id][0] == lane_id) {
                        comm[warp_id][1] = row_start; //vertex owned by the "lane_id"-th thread in a warp is scheduled
                        comm[warp_id][2] = row_end;
                        commr[warp_id] = ru;
                        commi[warp_id][1] = idxx;
                        row_start = row_end; //avoid processing this vertex repeatedly in <3>-while
                    }
                    size_t push_st = comm[warp_id][1] + lane_id; //process the "push_st"-th outgoing edge at first.
                    size_t push_ed = comm[warp_id][2];

                    // <2.1>-while: warp-granularity-outgoing edges
                    while (__any_sync(FULL_MASK, push_st<push_ed)) {
                        if (push_st < push_ed) {
                            v = csr_e[push_st];
                            testx = commi[warp_id][1] + csr_idx[push_st];
                            msg = commr[warp_id]*csr_w[testx];
                            if(v<vert_num){
                                atomicAdd(messages + v, msg);
                                isactive[v] = true;
                            }else{
                                int cur = atomicAdd(active_vert_num2,2);
                                active_vert2[cur]=v;
                                active_vert2[cur+1]=testx;
                                active_w2[cur/2] = commr[warp_id];
                            }
                        }
                        push_st += THREADS_PER_WARP; //until all outgoing edges of "u" have been processed
                    }
                } //until all source vertices with "todo_edges_num>=THREADS_PER_WARP" have been processed

                //while(3) then, the out-degree of "u" is less than THREADS_PER_WARP(32)
                int thread_count = row_end - row_start;
                int deg = thread_count;
                int scatter = 0, total = 0;

                __syncthreads();
                BlockScan(block_temp_storage).ExclusiveSum(thread_count, scatter, total); //
                __syncthreads(); //there are "total" edges left in every block
                int progress = 0;
                while (progress < total) {
                    int remain = total - progress;
                    while (scatter<(progress+THREADS_PER_BLOCK) && (row_start<row_end)) {
                        comm2[scatter-progress] = row_start;//存U有的外邻居
                        commd2[scatter-progress] = deg; //
                        commr2[scatter-progress] = ru;
                        commi2[scatter - progress][1] = idxx;
                        scatter++;
                        row_start++;
                    }
                    __syncthreads();
                    int cur_batch_count = min(remain, (int)THREADS_PER_BLOCK); //how many threads are required?
                    if (thread_id < cur_batch_count) {
                        v = csr_e[comm2[thread_id]];
                        testx = commi2[thread_id][1] + csr_idx[comm2[thread_id]];
                        msg = (commr2[thread_id]) *csr_w[testx];
                        if(v<vert_num){
                            atomicAdd(messages + v, msg);
                            isactive[v] = true;
                        }else{
                            int cur = atomicAdd(active_vert_num2,2);
                            active_vert2[cur]=v;
                            active_vert2[cur+1]=testx;
                            active_w2[cur/2] =commr2[thread_id];
                        }
                    }
                    __syncthreads();
                    progress += THREADS_PER_BLOCK;
                }
                //schedule (blockDim.x * gridDim.x) active vertices per <0>-while loop
                schedule_offset += blockDim.x * gridDim.x;
            }
            __syncthreads();
            __threadfence();
            if (threadIdx.x == 0) {
                atomicAdd((int *) &g_mutex1, 1);
                while (g_mutex1 < gridDim.x * iteration_id[l_iteration_id]) {}
            }
            __syncthreads();
            //host 有一个操作，将 active_verts_num 设置为0
            *active_vert_num = 0;
            total_avtive_num= *active_vert_num2/2;
            __syncthreads();
            __threadfence();


            schedule_offset = blockDim.x * blockIdx.x;
            while (schedule_offset < total_avtive_num) {
                idx = schedule_offset + thread_id;
                if (idx < total_avtive_num) {
                    u = active_vert2[idx*2];
                    idxx=active_vert2[idx*2+1];
                    ru = active_w2[idx];
                    row_start = csr_v[u]; //start offset of outgoing edges of "u" in "col_ind"
                    row_end = csr_v[u+1]; //end offset of outgoing edges of "u" in "col_ind" (exclusive)
                } else {
                    row_start = 0;
                    row_end = 0;
                }
                //while(1)
                while (__syncthreads_or((row_end-row_start)>=THREADS_PER_BLOCK)) {
                    if ((row_end-row_start) >= THREADS_PER_BLOCK) {
                        comm[0][0] = thread_id; //I (thread_id) want to process the active vertex assigned to me.
                    }
                    __syncthreads(); //all threads in one block vote to processing their own vertices

                    if (comm[0][0] == thread_id) {
                        comm[0][1] = row_start; //the vertx owned by me will be processed in this <1>-while loop.
                        comm[0][2] = row_end;
                        commr[0] = ru;//ru是u的残差
                        commi[0][1] = idxx;
                        row_start = row_end; //avoid processing this vertex repeatedly in <2>&<3>-while
                    }
                    __syncthreads(); //all threads are ready to process the selected vertex

                    size_t push_st = comm[0][1] + thread_id; //process the "push_st"-th outgoing edge at first.
                    size_t push_ed = comm[0][2];

                    // <1.1>-while: block-granularity-outgoing edges
                    while (__syncthreads_or(push_st<push_ed)) {
                        if (push_st < push_ed) {
                            v = csr_e[push_st]; //target vertex id
                            testx = commi[0][1] + csr_idx[push_st];
                            msg = commr[0]*csr_w[testx];
                            if(v<vert_num){
                                atomicAdd(messages + v, msg);
                                isactive[v] = true;
                            }else{
                                int cur = atomicAdd(active_vert_num1,2);
                                active_vert1[cur]=v;
                                active_vert1[cur+1]=testx;
                                active_w1[cur/2] = commr[0];
                            }
                        }
                        push_st += THREADS_PER_BLOCK; //until all outgoing edges of "u" have been processed
                    }
                } //until all source vertices with "todo_edges_num>=THREADS_PER_BLOCK" have been processed
                //while(2)
                while (__any_sync(FULL_MASK, (row_end-row_start)>=THREADS_PER_WARP)) {
                    if ((row_end-row_start) >= THREADS_PER_WARP) {
                        comm[warp_id][0] = lane_id; //threads in the "warp_id"-th warp try to vote
                    }
                    if (comm[warp_id][0] == lane_id) {
                        comm[warp_id][1] = row_start; //vertex owned by the "lane_id"-th thread in a warp is scheduled
                        comm[warp_id][2] = row_end;
                        commr[warp_id] = ru;
                        commi[warp_id][1] = idxx;
                        row_start = row_end; //avoid processing this vertex repeatedly in <3>-while
                    }
                    size_t push_st = comm[warp_id][1] + lane_id; //process the "push_st"-th outgoing edge at first.
                    size_t push_ed = comm[warp_id][2];

                    // <2.1>-while: warp-granularity-outgoing edges
                    while (__any_sync(FULL_MASK, push_st<push_ed)) {
                        if (push_st < push_ed) {
                            v = csr_e[push_st];
                            testx = commi[warp_id][1] + csr_idx[push_st];
                            msg = commr[warp_id]*csr_w[testx];
                            if(v<vert_num){
                                atomicAdd(messages + v, msg);
                                isactive[v] = true;
                            }else{
                                int cur = atomicAdd(active_vert_num1,2);
                                active_vert1[cur]=v;
                                active_vert1[cur+1]=testx;
                                active_w1[cur/2] = commr[warp_id];
                            }
                        }
                        push_st += THREADS_PER_WARP; //until all outgoing edges of "u" have been processed
                    }
                } //until all source vertices with "todo_edges_num>=THREADS_PER_WARP" have been processed

                //while(3) then, the out-degree of "u" is less than THREADS_PER_WARP(32)
                int thread_count = row_end - row_start;
                int deg = thread_count;
                int scatter = 0, total = 0;

                __syncthreads();
                BlockScan(block_temp_storage).ExclusiveSum(thread_count, scatter, total); //
                __syncthreads(); //there are "total" edges left in every block
                int progress = 0;
                while (progress < total) {
                    int remain = total - progress;
                    while (scatter<(progress+THREADS_PER_BLOCK) && (row_start<row_end)) {
                        comm2[scatter-progress] = row_start;//存U有的外邻居
                        commd2[scatter-progress] = deg; //
                        commr2[scatter-progress] = ru;
                        commi2[scatter - progress][1] = idxx;
                        scatter++;
                        row_start++;
                    }
                    __syncthreads();
                    int cur_batch_count = min(remain, (int)THREADS_PER_BLOCK); //how many threads are required?
                    if (thread_id < cur_batch_count) {
                        v = csr_e[comm2[thread_id]];
                        testx = commi2[thread_id][1] + csr_idx[comm2[thread_id]];
                        msg = (commr2[thread_id]) *csr_w[testx];
                        if(v<vert_num){
                            atomicAdd(messages + v, msg);
                            isactive[v] = true;
                        }else{
                            int cur = atomicAdd(active_vert_num1,2);
                            active_vert1[cur]=v;
                            active_vert1[cur+1]=testx;
                            active_w1[cur/2] =commr2[thread_id];
                        }
                    }
                    __syncthreads();
                    progress += THREADS_PER_BLOCK;
                }
                //schedule (blockDim.x * gridDim.x) active vertices per <0>-while loop
                schedule_offset += blockDim.x * gridDim.x;
            }
            __syncthreads();



            __threadfence();
            if (threadIdx.x == 0) {
                atomicAdd((int*) &g_mutex2, 1);
                while (g_mutex2 < gridDim.x * iteration_id[l_iteration_id]) {}

            }
            __syncthreads();

            //barrier 将新的活跃顶点加入边界
            volatile __shared__ size_t output_cta_offset;
            size_t thread_idx = threadIdx.x;
            size_t schedule_offset_barrir = blockDim.x * blockIdx.x;
            while (__syncthreads_or(schedule_offset_barrir < vert_num)) {
                vid = schedule_offset_barrir + thread_idx;
                int thread_cnt = 0;
                if (vid < vert_num) {
                    if (isactive[vid]) {
                        residual[vid] += messages[vid];
                        messages[vid] = 0;
                        isactive[vid] = false;
                        if ((residual[vid]) > (csr_ov[vid+1] - csr_ov[vid])*rmax) {//执行边界检测标准，符合条件将标志位设>置为1
                            pagerank[vid] += alpha * residual[vid];
                            thread_cnt = 1;
                        }
                    }
                }
                int scatter = 0, total = 0;

                __syncthreads();
                BlockScan(block_temp_storage).ExclusiveSum(thread_cnt, scatter, total);
                __syncthreads();
                if (thread_id == 0) {
                    output_cta_offset = atomicAdd(active_vert_num, total); //run per block
                }
                __syncthreads();
                if (thread_cnt > 0) {
                    active_vert[output_cta_offset+scatter] = vid;
                }
                schedule_offset_barrir += blockDim.x * gridDim.x;
            }

            __syncthreads();
            __threadfence();
            if (thread_id == 0) {
                atomicAdd((int*) &g_mutex4, 1);
                while (g_mutex4 < gridDim.x * iteration_id[l_iteration_id]) {}

            }
            __syncthreads();

            *active_vert_num2 = 0;

            __threadfence();
            if (threadIdx.x == 0) {
                atomicAdd((int *) &g_mutex5, 1);
                while (g_mutex5 < gridDim.x * iteration_id[l_iteration_id]) {}
            }

            __threadfence();
        }

        __threadfence();
        __syncthreads();
    }


}



