#include "hip/hip_runtime.h"
#include <time.h>
#include "Util.cuh"
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "Graph.h"
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>


__device__ volatile int g_mutex1;
__device__ volatile int g_mutex2;
__device__ volatile int g_mutex3;
__device__ volatile int g_mutex4;
__device__ volatile int g_mutex5;

__global__ void calcuatePPR(const ll *csr_v, const Vertex *csr_e, const ValueType *csr_w,
                            bool *f1, Vertex *act, Vertex *act_num, ValueType *pi, ValueType *oldval, ValueType *newval,
                            int *iteration_id,int *maxiter, int *record,
                            const int vert_num,const int source,const ValueType alpha
);


int main(int argc, char **argv) {
    // Initialize graph data in host & device memory
    hipFree(0);
    // 获取命令行参数
    std::string dir = argv[1];

    // 获取，csr_v ,csr_e ,v_r,degree,order;
    Graph graph(dir);
    std::cout << "test for study how to use cuda" << endl;
    int vert_num = graph.vert_num;
    int edge_num = graph.edge_num;
    int source = 1;

    thrust::device_vector<ll> d_csr_v(vert_num+1, 0);
    thrust::device_vector<Vertex> d_csr_e(edge_num,0);
    thrust::device_vector<ValueType> d_csr_w(edge_num,0);

    thrust::device_vector<bool> d_f1(vert_num, false);
    thrust::device_vector<Vertex> d_act(vert_num, 0);
    thrust::device_vector<Vertex> d_f(1,source);

    thrust::device_vector<ValueType> d_pi(vert_num,0.0);
   // thrust::host_vector<ValueType> h_pi(vert_num,0.0);
    thrust::device_vector<ValueType> d_oldval(vert_num,0);
    thrust::device_vector<ValueType> d_newval(vert_num,0);

    thrust::copy(graph.csr_v.begin(), graph.csr_v.end(), d_csr_v.begin());
    thrust::copy(graph.csr_e.begin(), graph.csr_e.end(), d_csr_e.begin());
    thrust::copy(graph.csr_w.begin(), graph.csr_w.end(), d_csr_w.begin());


    int *iteration_id;
    hipMalloc(&iteration_id, sizeof(int)*500);

    int *record;
    hipMalloc(&record, sizeof(int)*500);

    int *maxiter;
    hipMalloc(&maxiter, sizeof(int)*1);

    struct timeval t_start, t_stop;
    double timeuse;
    gettimeofday(&t_start, NULL);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    cout << "\n==================== PPR with FORWARD PUSH starts ====================" << endl;


    int cnt = 0;
    while(1){

        cnt++;
        calcuatePPR<<<MAX_BLOCKS_NUM, THREADS_PER_BLOCK>>>(
                d_csr_v.data().get(), d_csr_e.data().get(), d_csr_w.data().get(),
                d_f1.data().get(), d_act.data().get(), d_f.data().get(),d_pi.data().get(), d_oldval.data().get(), d_newval.data().get(),
                iteration_id,maxiter,record,
                vert_num, source,alpha
        );

        if(cnt==1){
            break;
        }
    }


    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float runtime = 0; //milliseconds
    hipEventElapsedTime(&runtime, start, stop);

    cout << "gpu runtime: " << runtime/1000.0 << " seconds" << endl;
    cout << "==================== PPR with FORWARD PUSH ends ====================\n" << endl;
    int *iteration_num = new int[1];
    int *iteration_act_num = new int[500];
    ValueType *h_pi = new ValueType[500];

    CUDA_ERROR(hipMemcpy(iteration_num, maxiter, sizeof(int)*1, hipMemcpyDeviceToHost));
    cout << "flag 已设置成 -1  终止条件以满足		iteration_num："<<iteration_num[0]<<endl;
    CUDA_ERROR(hipMemcpy(iteration_act_num, record, sizeof(int)*500, hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(h_pi, d_pi.data().get(), sizeof(ValueType)*500, hipMemcpyDeviceToHost));

    cout << "0	act_num：1"<<endl;
    for(int i = 1 ;i<50;i++){
        if(iteration_act_num[i]==0) break;
        cout <<i<< "	act_num："<<iteration_act_num[i]<<"---"<<h_pi[i]<<endl;

    }


    //cout << "内存开辟耗时: " << timeMalloc << endl;
    gettimeofday(&t_stop, NULL);

    timeuse = (t_stop.tv_sec - t_start.tv_sec) + (double)(t_stop.tv_usec - t_start.tv_usec)/1000000.0;
    //cout << "main total timeval runtime: " << timeuse << " seconds" << endl;
    return 0;
}



__global__ void calcuatePPR(const ll *csr_v, const Vertex *csr_e, const ValueType *csr_w,
                            bool *f1, Vertex *act, Vertex *act_num, ValueType *pi, ValueType *oldval, ValueType *newval,
                            int *iteration_id,int *maxiter, int *record,
                            const int vert_num,const int source,const ValueType alpha
){
    size_t thread_id = threadIdx.x;
    size_t schedule_offset_init = blockDim.x * blockIdx.x;
    size_t vid = 0;
    while (schedule_offset_init < vert_num) {
        vid = schedule_offset_init + thread_id;
        if(vid<vert_num){
            pi[vid]=0.0;
            newval[vid]=0.0;
            f1[vid]=false;
          //  if(vid<=source){
             //   oldval[vid]=10.0;
             //   act[vid]=vid;
           // }else{
                oldval[vid]=0.0;
           // }
        }
        if (vid < 500) {
            iteration_id[vid] = vid;
            record[vid]=0;
        }
        schedule_offset_init += blockDim.x * gridDim.x;
    }
    //prepare for the 1st iteration
    size_t global_id = thread_id + blockDim.x*blockIdx.x;
    if (global_id == 0) {
        *act_num=1;
        oldval[source]=10.0;
        act[0]=source;
        g_mutex1 = 0;
        g_mutex2 = 0;
        g_mutex4 = 0;
        g_mutex5 = 0;
        //printf("source =  初始化完成\n");
    }

    __syncthreads();
    __threadfence();
    if (threadIdx.x == 0) {
        atomicAdd((int*) &g_mutex3, 1);
        while ((g_mutex3 == 0) || (g_mutex3 % gridDim.x) ) {}
    }
    __syncthreads();

    int l_iteration_id = 0;
    int total_active_verts_num = *act_num;
    size_t lane_id = thread_id % THREADS_PER_WARP;
    size_t warp_id = thread_id / THREADS_PER_WARP; //the i-th warp (from 0)  当前块内warp的id
    typedef hipcub::BlockScan<int, THREADS_PER_BLOCK> BlockScan;
    __shared__ typename BlockScan::TempStorage block_temp_storage;
    volatile __shared__ int comm[THREADS_PER_BLOCK/THREADS_PER_WARP][3];//[256/32][3]第一维是多少个warp，256/32=8，8个warp，
    volatile __shared__ ValueType commr[THREADS_PER_BLOCK/THREADS_PER_WARP];//每个warp对应一个值
    volatile __shared__ int comm2[THREADS_PER_BLOCK]; //一维数组大小256，int
    volatile __shared__ ValueType commd2[THREADS_PER_BLOCK]; //out-degree
    volatile __shared__ ValueType commr2[THREADS_PER_BLOCK];

    while (total_active_verts_num>0&&l_iteration_id<400) {
        __syncthreads();
        l_iteration_id += 1;

        __syncthreads();
        __threadfence();
        if (thread_id == 0) {
            atomicAdd((int*) &g_mutex4, 1);
            while (g_mutex4 < gridDim.x * iteration_id[l_iteration_id]) {}

        }
        __syncthreads();
        size_t schedule_offset = blockDim.x * blockIdx.x;
        size_t idx = 0;
        int u = 0,v=0,w=0;
        int row_start, row_end;
        ValueType ru, msg;
        while (schedule_offset < total_active_verts_num) {
            idx = schedule_offset + thread_id;
            if (idx < total_active_verts_num) {
                u = act[idx];
                ru=oldval[u];
                row_start = csr_v[u]; //start offset of outgoing edges of "u" in "col_ind"
                row_end = csr_v[u + 1]; //end offset of outgoing edges of "u" in "col_ind" (exclusive)
            } else {
                row_start = 0;
                row_end = 0;
            }
            //while(1)
            while (__syncthreads_or((row_end - row_start) >= THREADS_PER_BLOCK)) {
                if ((row_end - row_start) >= THREADS_PER_BLOCK) {
                    comm[0][0] = thread_id; //I (thread_id) want to process the active vertex assigned to me.
                }
                __syncthreads(); //all threads in one block vote to processing their own vertices

                if (comm[0][0] == thread_id) {
                    comm[0][1] = row_start; //the vertx owned by me will be processed in this <1>-while loop.
                    comm[0][2] = row_end;
                    commr[0] = ru;//ru是u的残差
                    row_start = row_end; //avoid processing this vertex repeatedly in <2>&<3>-while
                }
                __syncthreads(); //all threads are ready to process the selected vertex

                size_t push_st = comm[0][1] + thread_id; //process the "push_st"-th outgoing edge at first.
                size_t push_ed = comm[0][2];

                // <1.1>-while: block-granularity-outgoing edges
                while (__syncthreads_or(push_st < push_ed)) {
                    if (push_st < push_ed) {
                        v = csr_e[push_st]; //target vertex id
                        msg=csr_w[push_st]*commr[0];
                        atomicAdd(newval + v, msg);
                        f1[v]=true;


                    }
                    push_st += THREADS_PER_BLOCK; //until all outgoing edges of "u" have been processed
                }
            } //until all source vertices with "todo_edges_num>=THREADS_PER_BLOCK" have been processed


            //while(2)
            while (__any_sync(FULL_MASK, (row_end-row_start)>=THREADS_PER_WARP)) {
                if ((row_end-row_start) >= THREADS_PER_WARP) {
                    comm[warp_id][0] = lane_id; //threads in the "warp_id"-th warp try to vote
                }
                if (comm[warp_id][0] == lane_id) {
                    comm[warp_id][1] = row_start; //vertex owned by the "lane_id"-th thread in a warp is scheduled
                    comm[warp_id][2] = row_end;
                    commr[warp_id] = ru;
                    row_start = row_end; //avoid processing this vertex repeatedly in <3>-while
                }
                size_t push_st = comm[warp_id][1] + lane_id; //process the "push_st"-th outgoing edge at first.
                size_t push_ed = comm[warp_id][2];

                // <2.1>-while: warp-granularity-outgoing edges
                while (__any_sync(FULL_MASK, push_st<push_ed)) {
                    if (push_st < push_ed) {
                        v = csr_e[push_st];
                        msg=csr_w[push_st]*commr[warp_id];
                        atomicAdd(newval + v, msg);
                        f1[v]=true;
                    }
                    push_st += THREADS_PER_WARP; //until all outgoing edges of "u" have been processed
                }
            } //until all source vertices with "todo_edges_num>=THREADS_PER_WARP" have been processed

            //while(3) then, the out-degree of "u" is less than THREADS_PER_WARP(32)
            int thread_count = row_end - row_start;
            int deg = thread_count;
            int scatter = 0, total = 0;

            __syncthreads();
            BlockScan(block_temp_storage).ExclusiveSum(thread_count, scatter, total); //
            __syncthreads(); //there are "total" edges left in every block
            int progress = 0;

            while (progress < total) {
                int remain = total - progress;
                while (scatter<(progress+THREADS_PER_BLOCK) && (row_start<row_end)) {
                    comm2[scatter-progress] = row_start;//存U有的外邻居
                    commr2[scatter-progress] = ru;
                    scatter++;
                    row_start++;
                }
                __syncthreads();
                int cur_batch_count = min(remain, (int)THREADS_PER_BLOCK); //how many threads are required?
                if (thread_id < cur_batch_count) {
                    v = csr_e[comm2[thread_id]];
                    msg=csr_w[comm2[thread_id]]*commr2[thread_id];
                    atomicAdd(newval + v, msg);
                    f1[v]=true;
                }
                __syncthreads();
                progress += THREADS_PER_BLOCK;
            }
            //schedule (blockDim.x * gridDim.x) active vertices per <0>-while loop
            schedule_offset += blockDim.x * gridDim.x;
        }

        __syncthreads();
        __threadfence();
        if (threadIdx.x == 0) {
            atomicAdd((int*) &g_mutex5, 1);
            while (g_mutex5 < gridDim.x * iteration_id[l_iteration_id]) {}

        }
        __syncthreads();
       *act_num = 0;
        __syncthreads();
        __threadfence();
 
        if (threadIdx.x == 0) {
            atomicAdd((int*) &g_mutex1, 1);
            while (g_mutex1 < gridDim.x * iteration_id[l_iteration_id]) {}

        }
        __syncthreads();

        //barrier 将新的活跃顶点加入边界
        volatile __shared__ size_t output_cta_offset;
        size_t thread_idx = threadIdx.x;
        size_t schedule_offset_barrir = blockDim.x * blockIdx.x;
        size_t vid = 0;
        ValueType delta=0.0;
        while (__syncthreads_or(schedule_offset_barrir < vert_num)) {
            vid = schedule_offset_barrir + thread_idx;
            int thread_cnt = 0;
            if (vid < vert_num) {
                if (f1[vid]) {
                    f1[vid] = false;
                    if (newval[vid]/(csr_v[vid+1]-csr_v[vid]) > 0.0001) {//执行边界检测标准，符合条件将标志位设>置为1
                        delta=alpha*newval[vid];
                        newval[vid]=0.0;
                        oldval[vid]=delta;
                        pi[vid]+=delta;
                        thread_cnt = 1;
                    }
                }
            }
            int scatter = 0, total = 0;

            __syncthreads();
            BlockScan(block_temp_storage).ExclusiveSum(thread_cnt, scatter, total);
            __syncthreads();
            if (thread_id == 0) {
                output_cta_offset = atomicAdd(act_num, total); //run per block
            }
            __syncthreads();
            if (thread_cnt > 0) {
                act[output_cta_offset+scatter] = vid;
            }
            schedule_offset_barrir += blockDim.x * gridDim.x;
        }

        __syncthreads();
        __threadfence();
        if (thread_id == 0) {
            atomicAdd((int*) &g_mutex2, 1);
            while (g_mutex2 < gridDim.x * iteration_id[l_iteration_id]) {}
            maxiter[0] = l_iteration_id;
            if(l_iteration_id<500)	record[l_iteration_id]= *act_num;

        }
        __syncthreads();
        __threadfence();
        total_active_verts_num = *act_num;
        __syncthreads();

    }



}

