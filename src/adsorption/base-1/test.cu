#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cout << "No CUDA-capable devices found." << std::endl;
        return 1;
    }

    for (int dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout << "Device " << dev << ": " << deviceProp.name << std::endl;
        std::cout << "  Total number of multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "  Total number of threads per multiprocessor: " << deviceProp.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "  Total number of threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
    }

    return 0;
}